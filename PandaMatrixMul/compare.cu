
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 
	
	File: compare.cu 
	First Version:		2012-07-01 V0.1
	Current Version:	2012-09-01 V0.3	
	Last Updates:		2012-09-02

	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */

#ifndef __COMPARE_CU__
#define __COMPARE_CU__
#include "Panda.h"
#include "Global.h"


//-----------------------------------------------------------
//No Sort in this application
//-----------------------------------------------------------
__device__ int compare(const void *d_a, int len_a, const void *d_b, int len_b)
{


	return 0;
}


int cpu_compare(const void *d_a, int len_a, const void *d_b, int len_b)
{


	return 0;
}


#endif //__COMPARE_CU__
