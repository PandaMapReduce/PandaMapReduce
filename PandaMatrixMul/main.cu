
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 
	
	File: main.cu 
	First Version:		2012-07-01 V0.1
	Current Version:	2012-09-01 V0.3	
	Last Updates:		2012-09-02

	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */


#include "Panda.h"
#include "UserAPI.h"
#include <ctype.h>

//-----------------------------------------------------------------------
//	Panda Matrix Multiplication
//-----------------------------------------------------------------------

static float *GenMatrix(int M_ROW_COUNT, int M_COL_COUNT, float init)
{
        float *matrix = (float*)malloc(sizeof(float)*M_ROW_COUNT*M_COL_COUNT);
        //srand(time(0));
		//memset(matrix,1.0,sizeof(float)*M_ROW_COUNT*M_COL_COUNT);
		for (int i = 0; i < M_ROW_COUNT; i++)
                for (int j = 0; j < M_COL_COUNT; j++)
                        //matrix[i*M_COL_COUNT+j] = (float)(rand() % 100);
						matrix[i*M_COL_COUNT+j] = init;
		return matrix;
}

static float *RotateMatrix(float *matrix, int rowCount, int colCount)
{
        float *m = (float*)malloc(sizeof(float)*rowCount*colCount);
        for (int i = 0; i < rowCount; i++)
                for (int j = 0; j < colCount; j++)
                                m[i * colCount + j] = matrix[i + colCount * j];
        return m;
}//static float
		
int main(int argc, char** argv)
{		
	if (argc != 5)
	{	
		printf("usage: %s [matrix size][num gpus][num cpu groups][cpu/gpu work ratio]\n", argv[0]);
		exit(-1);	
	}//if
	
	ShowLog("configure input data for Panda job");

	int ROW_NUM = atoi(argv[1]);
	int COL_NUM = atoi(argv[1]);
	int num_gpus = atoi(argv[2]);
	int num_cpus_groups = atoi(argv[3]);
	int num_mappers = 1;//atoi(argv[5]);
	float ratio = atof(argv[4]);
	int auto_tune = 0;//atoi(argv[7]);

	double t1 = PandaTimer();

	float *matrix1 = GenMatrix(ROW_NUM,COL_NUM, 1.0);
	float *tmpMatrix2 = GenMatrix(COL_NUM,ROW_NUM, 1.0);
    float *matrix2 = RotateMatrix(tmpMatrix2,COL_NUM,ROW_NUM);
	float *matrix3 = GenMatrix(ROW_NUM,COL_NUM, 0.0);

	double t2 = PandaTimer();
	ShowLog("load matrices  num_gpus:%d  num_cpus_groups:%d", num_gpus, num_cpus_groups);
	double t3 = PandaTimer();

	MM_KEY_T key;
    MM_VAL_T val;
	val.row_dim = ROW_NUM;
    val.col_dim = COL_NUM;
	//val.mbz = MATRIX_BLOCK_SIZE;
	//val.tbz = THREAD_BLOCK_SIZE;

	int start_row_id, end_row_id;
	start_row_id = end_row_id = 0;

	int cpu_task_num = (int)(ROW_NUM*ratio);
	int gpu_task_num = ROW_NUM - cpu_task_num;
	//panda worker
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*(num_gpus + num_cpus_groups));

	
	for (int dev_id=0; dev_id<num_cpus_groups; dev_id++){
		if (cpu_task_num == 0)
			break;

		//panda job
		job_configuration *cpu_job_conf = CreateJobConf();
		cpu_job_conf->num_cpus_groups = num_cpus_groups;
		cpu_job_conf->num_cpus_cores = getCPUCoresNum();

		int partitions = cpu_job_conf->num_cpus_cores*num_cpus_groups;
		int task_per_partition = ((cpu_task_num)/partitions);

		key.h_matrix1 = matrix1;
		key.h_matrix2 = matrix2;
		key.h_matrix3 = matrix3;

		key.matrix1 = NULL;
		key.matrix2 = NULL;
		key.matrix3 = NULL;

		for (int i=0;i<partitions;i++){
			start_row_id = task_per_partition*i;
			end_row_id = start_row_id+task_per_partition;
			val.row = start_row_id;
			val.col = end_row_id;
			if( i == (partitions-1) )
				val.col = cpu_task_num;

			if (val.col > val.row)
				AddPandaTask(cpu_job_conf, &key, &val, sizeof(MM_KEY_T), sizeof(MM_VAL_T));
		}//for

		thread_info[dev_id].job_conf = cpu_job_conf;
		thread_info[dev_id].device_type = CPU_ACC;

	}//for
		
	for (int dev_id=0; dev_id<num_gpus; dev_id++){

		job_configuration *gpu_job_conf = CreateJobConf();
		gpu_job_conf->num_gpus = num_gpus;
		gpu_job_conf->num_mappers = num_mappers;
		gpu_job_conf->auto_tuning = false;
		gpu_job_conf->ratio = (double)ratio;
		gpu_job_conf->auto_tuning_sample_rate = -1;//sample_rate;

		if ( dev_id == 0 )
			start_row_id = cpu_task_num;	
		else 
			start_row_id = cpu_task_num+(dev_id)*(gpu_task_num/num_gpus);

		end_row_id = start_row_id + gpu_task_num/num_gpus;
		if ( dev_id == (num_gpus - 1) )
			end_row_id = cpu_task_num + gpu_task_num;

		//copy to data into different GPU device
		hipSetDevice(dev_id);  

		int matrixSize = sizeof(float)*ROW_NUM*COL_NUM;
		float *d_matrix1 = NULL;
		hipMalloc((void **)&d_matrix1,matrixSize);
		hipMemcpy(d_matrix1, matrix1, matrixSize, hipMemcpyHostToDevice);
		float *d_matrix2 = NULL;
		hipMalloc((void**)&d_matrix2,matrixSize);
		hipMemcpy(d_matrix2,matrix2,matrixSize,hipMemcpyHostToDevice);
		float *d_matrix3 = NULL;
		hipMalloc((void**)&d_matrix3,matrixSize);
		hipMemcpy(d_matrix3,matrix3,matrixSize,hipMemcpyHostToDevice);

		key.matrix1 = d_matrix1;
		key.matrix2 = d_matrix2;
		key.matrix3 = d_matrix3;

		key.h_matrix1 = NULL;
		key.h_matrix2 = NULL;
		key.h_matrix3 = NULL;

		if(end_row_id>start_row_id)
			for (int i = start_row_id/MATRIX_BLOCK_SIZE; i < (end_row_id + MATRIX_BLOCK_SIZE-1)/MATRIX_BLOCK_SIZE; i++)
			{
				val.row = i;
				for (int j = 0; j < (COL_NUM + MATRIX_BLOCK_SIZE-1)/MATRIX_BLOCK_SIZE; j++)
				{
					val.col = j;
					AddPandaTask(gpu_job_conf, &key, &val, sizeof(MM_KEY_T), sizeof(MM_VAL_T));
				}//for
			}//for
			thread_info[num_cpus_groups + dev_id].job_conf = gpu_job_conf;
			thread_info[num_cpus_groups + dev_id].device_type = GPU_ACC;
	}
	
	double t4 = PandaTimer();
	panda_context *panda = CreatePandaContext();
	
	panda->num_gpus = num_gpus;
	panda->num_cpus_groups = num_cpus_groups;
	panda->ratio = ratio;

	/*if (auto_tune ==1){
	ratio = Smart_Scheduler(thread_info, panda);
	panda->ratio = ratio;
	}
	else*/
	PandaMetaScheduler(thread_info, panda);
	
	hipDeviceSynchronize();
	double t5 = PandaTimer();

	ShowLog("GenMatrix:%f",t2-t1);
	ShowLog("Copy to GPU:%f",t4-t3);
	ShowLog("Compute:%f",t5-t4);
	char str[128];
	sprintf(str,"matrix size:%d copy2GPU:%f  compute:%f cpu/gpu ratio:%f auto-tune:%d", ROW_NUM, t3-t2, t5-t4, (double)ratio,auto_tune);
	DoDiskLog(str);

	return 0;
}//		