#include "hip/hip_runtime.h"
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 
	
	File: PandaSched.cu 
	First Version:		2012-07-01 V0.1
	Current Version:	2012-09-01 V0.3	
	Last Updates:		2012-09-02

	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */

#ifndef _PANDASCHED_CU_
#define _PANDASCHED_CU_

// includes, kernels
#include "Panda.h"
#include "UserAPI.h"


//--------------------------------------------------
//  PandaMetaScheduler
//--------------------------------------------------
/*
 * 1) input a set of panda worker (thread)
 * 2) each panda worker consist of one panda job and pand device
 * 3) copy input data from pand job to pand device 
 */



//For version 0.3
void PandaMetaScheduler(thread_info_t *thread_info, panda_context *panda){

	int num_gpus = panda->num_gpus;
	int num_cpus_groups = panda->num_cpus_groups;
	float ratio = panda->ratio;
	
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_groups));
	
	int assigned_gpu_id = 0;
	int assigned_cpu_group_id = 0;
	for (int dev_id=0; dev_id<(num_gpus + num_cpus_groups); dev_id++){

		if (thread_info[dev_id].device_type == GPU_ACC){
			
			gpu_context *d_g_state = CreateGPUContext();
			job_configuration *gpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);

			d_g_state->num_mappers = gpu_job_conf->num_mappers;
			d_g_state->num_reducers = gpu_job_conf->num_reducers;
			d_g_state->num_gpus = num_gpus;
			d_g_state->gpu_id = assigned_gpu_id;
			d_g_state->local_combiner = gpu_job_conf->local_combiner;
			d_g_state->iterative_support = gpu_job_conf->iterative_support;

			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;

			ShowLog("Assigned Dev_ID:[%d] GPU_ACC TID:%d",assigned_gpu_id,thread_info[dev_id].tid);
			assigned_gpu_id++;

		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
			
			cpu_context *d_g_state = CreateCPUContext();
			job_configuration *cpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
			
			d_g_state->cpu_group_id = assigned_cpu_group_id;
			d_g_state->local_combiner = cpu_job_conf->local_combiner;
			d_g_state->iterative_support = cpu_job_conf->iterative_support;

			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;

			ShowLog("Assigned Dev_ID:[%d] CPU_ACC TID:%d",dev_id,thread_info[dev_id].tid);
			assigned_cpu_group_id++;

		}//if
	}//for
	
	///////////////////////////////////////////////////
		
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){

		if (thread_info[dev_id].device_type == GPU_ACC){

				job_configuration *gpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_task_id = 0;
				int end_task_id = gpu_job_conf->num_input_record;
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				
				AddMapInputRecordGPU(d_g_state,(gpu_job_conf->input_keyval_arr), start_task_id,end_task_id);
				
		}//if
	
		if (thread_info[dev_id].device_type == CPU_ACC){

				job_configuration *cpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_task_id = 0;
				int end_task_id = cpu_job_conf->num_input_record;
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				
				//ShowWarn("cpu_job_conf->num_input_record:%d ",cpu_job_conf->num_input_record);
				AddMapInputRecordCPU(d_g_state,(cpu_job_conf->input_keyval_arr),start_task_id, end_task_id);
				
		}//if
	}//for

	//1) initial map; 2)run map task; 3) run local combiner
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
		if (pthread_create(&(no_threads[dev_id]), NULL, Panda_Map, (char *)&(thread_info[dev_id])) != 0) 
			perror("Thread creation failed!\n");
	}//for

	for (int i = 0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for
	
	
	for (int i = 0; i < num_gpus + num_cpus_groups; i++){

		if (thread_info[i].device_type == CPU_ACC){
			
			PandaShuffleMergeCPU((panda_context*)panda, (cpu_context*)(thread_info[i].d_g_state));
			//ShowLog("==>CPU d_g_state_1->sorted_keyvals_arr_len:%d",panda->sorted_keyvals_arr_len);
		}//if

		if (thread_info[i].device_type == GPU_ACC){
			
			PandaShuffleMergeGPU((panda_context*)panda, (gpu_context*)(thread_info[i].d_g_state));
			//ShowLog("==>GPU d_g_state_1->sorted_keyvals_arr_len:%d",panda->sorted_keyvals_arr_len);
		}//if
			
	}//for
	
	int num_sorted_intermediate_record = panda->sorted_keyvals_arr_len;
	int num_tasks_4_cpu = num_sorted_intermediate_record*(panda->ratio);
	int num_tasks_4_gpu = num_sorted_intermediate_record*(1 - panda->ratio);

	int task_per_cpu = 0;
	if (num_cpus_groups>0) task_per_cpu = num_tasks_4_cpu/(num_cpus_groups);

	int task_per_gpu = 0;
	if (num_gpus>0) task_per_gpu = num_tasks_4_gpu/(num_gpus);
	int *split = (int*)malloc(sizeof(int)*(num_gpus+num_cpus_groups));
	
	for (int i=0; i<num_gpus; i++){
		if (i==0) 	split[0] = task_per_gpu;
		else if (i== num_gpus -1) split[i] = num_tasks_4_gpu;
		else split[i] = split[i-1] + task_per_gpu;
		//ShowLog("i:%d  %d",i, split[i]);
	}//for
	
	for (int i=num_gpus; i<num_gpus+num_cpus_groups; i++){
		if (i==0) 	split[0] = task_per_cpu;
		else if (i == num_gpus + num_cpus_groups - 1) split[i] = num_tasks_4_gpu + num_tasks_4_cpu;
		else split[i] = split[i-1] + task_per_cpu;
		//ShowLog("i:%d  %d",i, split[i]);
	}//for
	split[num_gpus + num_cpus_groups - 1] = num_sorted_intermediate_record;

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
	
		int start_row_id = 0;
		if (dev_id>0) start_row_id = split[dev_id-1];
		int end_row_id = split[dev_id];
				
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				AddReduceInputRecordGPU(d_g_state,(panda->sorted_intermediate_keyvals_arr), start_row_id, end_row_id);
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				d_g_state->sorted_keyvals_arr_len = 0;
				AddReduceInputRecordCPU(d_g_state,(panda->sorted_intermediate_keyvals_arr), start_row_id, end_row_id);
		}//if

	}//for

	for (int dev_id = 0; dev_id < (num_gpus+num_cpus_groups); dev_id++){
		if (pthread_create(&(no_threads[dev_id]),NULL,Panda_Reduce,(char *)&(thread_info[dev_id]))!=0) 
			perror("Thread creation failed!\n");
	}//for
		
	for (int i=0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for

	//TODO Reduce Merge
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	//Panda_Reduce_Merge(&thread_info[num_gpus-1]);															 //
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	
	int total_output_records = 0;
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
	
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->d_reduced_keyval_arr_len;
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->sorted_keyvals_arr_len;
		}//if
		
	}//for

	ShowLog("number of reduce output:%d\n",total_output_records);
	ShowLog("=====panda mapreduce job finished=====");

}//PandaMetaScheduler


//Scheduler for version 0.2 depressed

//Ratio = Tcpu/Tgpu
//Tcpu = (execution time on CPU cores for sampled tasks)/(#sampled tasks)
//Tgpu = (execution time on 1 GPU for sampled tasks)/(#sampled tasks)
//smart scheduler for auto tuning; measure the performance of sample data  

float AutoTuningScheduler(thread_info_t *thread_info, panda_context *panda){
	
	ShowLog("AutoTuningScheduler");
	int num_gpus = panda->num_gpus;
	int num_cpus_cores = getCPUCoresNum();//job_conf->num_cpus;
	int num_cpus_groups = panda->num_cpus_groups;
	
	num_gpus = 1;
	num_cpus_groups = 1;

	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_groups));
	
	int cpu_sampled_tasks_num = 0;
	int gpu_sampled_tasks_num = 0;

	int start_row_id = 0;
	int end_row_id = 0;//job_conf->num_cpus_cores*2; //(job_conf->num_input_record/100);

	int cpu_index = -1;
	int gpu_index = -1;

	for (int tid=0; tid<num_gpus+num_cpus_groups; tid++){
	
		if (thread_info[tid].device_type == GPU_ACC){
			if (gpu_index>=0)
				continue;
			gpu_index = tid;

			gpu_context *d_g_state = CreateGPUContext();
			d_g_state->num_gpus = num_gpus;
			thread_info[tid].d_g_state = d_g_state;

			job_configuration *gpu_job_conf = (job_configuration *)(thread_info[tid].job_conf);
			gpu_sampled_tasks_num = gpu_job_conf->num_input_record;
			start_row_id = 0;
			end_row_id = gpu_job_conf->num_input_record;
			AddMapInputRecordGPU(d_g_state,(gpu_job_conf->input_keyval_arr), start_row_id, end_row_id);
			
		}//if
		
		if (thread_info[tid].device_type == CPU_ACC){
			if (cpu_index>=0)
				continue;
			cpu_index = tid;

			cpu_context *d_g_state = CreateCPUContext();
			//d_g_state->num_cpus_groups = num_cpus_groups;
			thread_info[tid].d_g_state = d_g_state;

			job_configuration *cpu_job_conf = (job_configuration *)(thread_info[tid].job_conf);
			cpu_sampled_tasks_num = cpu_job_conf->num_input_record;
			start_row_id = 0;
			end_row_id = cpu_job_conf->num_input_record;
			AddMapInputRecordCPU(d_g_state,(cpu_job_conf->input_keyval_arr), start_row_id, end_row_id);

		}//if
	}//for 
	
	//cpu_sampled_tasks_num = num_cpus_cores*job_conf->auto_tuning_sample_rate;
	//gpu_sampled_tasks_num = getGPUCoresNum()*job_conf->auto_tuning_sample_rate;
	//if (cpu_sampled_tasks_num>job_conf->num_input_record)
	//if (gpu_sampled_tasks_num>job_conf->num_input_record)
		
	double t1 = PandaTimer();
	Panda_Map((void *)&(thread_info[gpu_index]));
	double t2 = PandaTimer();
	//start_row_id cpu 
	Panda_Map((void *)&(thread_info[cpu_index]));
	double t3 = PandaTimer();
	
	double t_cpu = (t3-t2);///cpu_sampled_tasks_num;
	double t_gpu = (t2-t1);///gpu_sampled_tasks_num;

	if (t_gpu<0.0001)
		t_gpu=0.0001;
	
	//double ratio = (t_cpu*gpu_sampled_tasks_num)/(t_gpu*cpu_sampled_tasks_num);
	
	double ratio = (t_cpu)/(t_gpu);
	ShowLog("cpu time:%f gpu time:%f ratio:%f", (t_cpu), (t_gpu), ratio);
	/*
	char log[128];
	sprintf(log,"	cpu_sampled_tasks:%d cpu time:%f cpu time per task:%f", cpu_sampled_tasks_num, t_cpu, t_cpu/(cpu_sampled_tasks_num));
	DoDiskLog(log);
	sprintf(log,"	gpu_sampled_tasks:%d gpu time:%f gpu time per task:%f	ratio:%f", gpu_sampled_tasks_num, t_gpu, t_gpu/(gpu_sampled_tasks_num), ratio);
	DoDiskLog(log);
	*/
	
	return (ratio);
	
}//void

void PandaDynamicMetaScheduler(thread_info_t *thread_info, panda_context *panda){

	int num_gpus = panda->num_gpus;
	int num_cpus_groups = panda->num_cpus_groups;
	float ratio = panda->ratio;
	
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_groups));
	
	for (int dev_id=0; dev_id<(num_gpus + num_cpus_groups); dev_id++){

		int assigned_gpu_id = 0;
		if (thread_info[dev_id].device_type == GPU_ACC){
			
			job_configuration* gpu_job_conf = (job_configuration*)(thread_info[dev_id].job_conf);
			gpu_context *d_g_state = CreateGPUContext();
			d_g_state->num_mappers = gpu_job_conf->num_mappers;
			d_g_state->num_reducers = gpu_job_conf->num_reducers;
			d_g_state->num_gpus = num_gpus;
			d_g_state->gpu_id = assigned_gpu_id;

			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;

			ShowLog("Assigned Dev_ID:[%d] GPU_ACC TID:%d",assigned_gpu_id,thread_info[dev_id].tid);
			assigned_gpu_id++;
		}//if

		int cpu_group_id = 0;
		if (thread_info[dev_id].device_type == CPU_ACC){
			
			cpu_context *d_g_state = CreateCPUContext();
			d_g_state->cpu_group_id = cpu_group_id;
			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;

			ShowLog("Assigned Dev_ID:[%d] CPU_ACC TID:%d",dev_id,thread_info[dev_id].tid);
			cpu_group_id++;
		}//if
	}//for

	///////////////////////////////////////////////////
	
	
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){

		if (thread_info[dev_id].device_type == GPU_ACC){

				job_configuration *gpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_row_id = 0;
				int end_id = gpu_job_conf->num_input_record;
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);

				AddMapInputRecordGPU(d_g_state,(gpu_job_conf->input_keyval_arr), start_row_id,end_id);
				
		}//if
	
		if (thread_info[dev_id].device_type == CPU_ACC){

				job_configuration *cpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_row_id = 0;
				int end_id = cpu_job_conf->num_input_record;
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				
				AddMapInputRecordCPU(d_g_state,(cpu_job_conf->input_keyval_arr),start_row_id, end_id);
				
		}//if
	}//for
	
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
		if (pthread_create(&(no_threads[dev_id]), NULL, Panda_Map, (char *)&(thread_info[dev_id])) != 0) 
			perror("Thread creation failed!\n");
	}//for

	for (int i = 0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for

	//ShowLog("start to merge results of GPU's and CPU's device to Panda scheduler");
	for (int i = 0; i < num_gpus+num_cpus_groups; i++){

		if (thread_info[i].device_type == CPU_ACC)
			PandaShuffleMergeCPU((panda_context*)panda, (cpu_context*)(thread_info[i].d_g_state));

		if (thread_info[i].device_type == GPU_ACC)
			
			((panda_context*)panda, (gpu_context*)(thread_info[i].d_g_state));
			
	}//for
	
	//TODO reduce task ratio 
	int num_sorted_intermediate_record = panda->sorted_keyvals_arr_len;
	int records_per_device = num_sorted_intermediate_record/(num_gpus + num_cpus_groups*ratio);
	
	int *split = (int*)malloc(sizeof(int)*(num_gpus+num_cpus_groups));
	
	for (int i=0; i<num_gpus; i++){
	
				if (i==0) 
				split[0] = records_per_device;
				else
				split[i] = split[i-1] + records_per_device;
				
	}//for
	
	for (int i=num_gpus; i<num_gpus+num_cpus_groups; i++){
	
				if (i==0) 
				split[0] = records_per_device*ratio;
				else 
				split[i] = split[i-1] + records_per_device*ratio;
								
	}//for
	split[num_gpus + num_cpus_groups-1] = num_sorted_intermediate_record;

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
	
		int start_row_id = 0;
		if (dev_id>0) start_row_id = split[dev_id-1];
		int end_row_id = split[dev_id];
				
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				ShowLog("start:%d end:%d",start_row_id,end_row_id);
				AddReduceInputRecordGPU(d_g_state,(panda->sorted_intermediate_keyvals_arr), start_row_id, end_row_id);
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				ShowLog("start:%d end:%d",start_row_id,end_row_id);
				AddReduceInputRecordCPU(d_g_state,(panda->sorted_intermediate_keyvals_arr), start_row_id, end_row_id);
		}//if
	}//for

	for (int dev_id = 0; dev_id < (num_gpus+num_cpus_groups); dev_id++){
		if (pthread_create(&(no_threads[dev_id]),NULL,Panda_Reduce,(char *)&(thread_info[dev_id]))!=0) 
			perror("Thread creation failed!\n");
	}//for
		
	for (int i=0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for

	//TODO Reduce Merge
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	//Panda_Reduce_Merge(&thread_info[num_gpus-1]);															 //
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	
	int total_output_records = 0;
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
	
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->d_reduced_keyval_arr_len;
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->sorted_keyvals_arr_len;
		}//if
		
	}//for
	ShowLog("number of reduce output:%d\n",total_output_records);
	ShowLog("=====panda mapreduce job finished=====");

}//PandaMetaScheduler


#endif // _PRESCHED_CU_
