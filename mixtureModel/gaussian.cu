#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>
#include "gaussian.h"
#include "invert_matrix.h"

// includes, kernels
#include <theta_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest( int argc, char** argv);

extern "C"
float* readData(char* f, int* ndims, int*nevents);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) {
    runTest( argc, argv);

    //CUT_EXIT(argc, argv);
}

///////////////////////////////////////////////////////////////////////////////
// Validate command line arguments
///////////////////////////////////////////////////////////////////////////////
int validateArguments(int argc, char** argv, int* num_clusters) {
    if(argc <= 4 && argc >= 3) {
        // parse num_clusters
        if(!sscanf(argv[1],"%d",num_clusters)) {
            printf("Invalid number of starting clusters\n\n");
            printUsage(argv);
            return 1;
        } 
        
        // Check bounds for num_clusters
        if(*num_clusters < 1 || *num_clusters > MAX_CLUSTERS) {
            printf("Invalid number of starting clusters\n\n");
            printUsage(argv);
            return 1;
        }
        
        // parse infile
        FILE* infile = fopen(argv[2],"r");
        if(!infile) {
            printf("Invalid infile.\n\n");
            printUsage(argv);
            return 2;
        } 
        
        // parse outfile
        if(argc == 4) {
            FILE* outfile = fopen(argv[3],"w");
            if(!outfile) {
                printf("Unable to create output file.\n\n");
                printUsage(argv);
                return 3;
            }
            fclose(outfile);
        }
        
        // Clean up so the EPA is happy
        fclose(infile);
        return 0;
    } else {
        printUsage(argv);
        return 1;
    }
}

///////////////////////////////////////////////////////////////////////////////
// Print usage statement
///////////////////////////////////////////////////////////////////////////////
void printUsage(char** argv)
{
   printf("Usage: %s num_clusters infile [outfile]\n",argv[0]);
   printf("\t num_clusters: The number of starting clusters\n");
   printf("\t infile: ASCII space-delimited FCS data file\n");
   printf("\t outfile: Clustering results output file\n");
}

void printCluster(cluster c, int num_dimensions) {
    printf("Probability: %f\n", c.pi);
    printf("N: %f\n",c.N);
    printf("Spectral Mean: ");
    for(int i=0; i<num_dimensions; i++){
        printf("%.3f ",c.means[i]);
    }
    printf("\n");

    printf("\nR Matrix:\n");
    for(int i=0; i<num_dimensions; i++) {
        for(int j=0; j<num_dimensions; j++) {
            printf("%.3f ", c.R[i*num_dimensions+j]);
        }
        printf("\n");
    }   
    
    printf("\nR-inverse Matrix:\n");
    for(int i=0; i<num_dimensions; i++) {
        for(int j=0; j<num_dimensions; j++) {
            printf("%.3f ", c.Rinv[i*num_dimensions+j]);
        }
        printf("\n");
    } 
    
    printf("\nConstant: %f\n",c.constant);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
int
runTest( int argc, char** argv) 
{
    
    int num_clusters;
    
    int error = validateArguments(argc,argv,&num_clusters);
    
    // Don't continue if we had a problem with the program arguments
    if(error) {
        return 1;
    }
    
    int num_dimensions;
    int num_events;
    
    // Read FCS data    
    float* fcs_data = readData(argv[2],&num_dimensions,&num_events);
    
    if(!fcs_data) {
        printf("Error parsing input file. This could be due to an empty file ");
        printf("or an inconsistent number of dimensions. Aborting.\n");
        return 1;
    }
    
    printf("Number of events: %d\n",num_events);
    printf("Number of dimensions: %d\n\n",num_dimensions);
    
    CUT_DEVICE_INIT(argc, argv);

    
    // print the input
    for( unsigned int i = 0; i < num_events*num_dimensions; i += num_dimensions ) 
    {
        for(unsigned int j = 0; j < num_dimensions; j++) {
            //printf("%f ",fcs_data[i+j]);
        }
        //printf("\n");
    }
    
    unsigned int num_threads = num_dimensions*num_dimensions;
    if(num_threads > NUM_THREADS) {
        num_threads = NUM_THREADS;
    }

    // Setup the cluster data structures on host
    cluster* clusters = (cluster*)malloc(sizeof(cluster)*num_clusters);
    for(int i=0; i<num_clusters;i++) {
        clusters[i].N = 0.0;
        clusters[i].pi = 0.0;
        clusters[i].means = (float*) malloc(sizeof(float)*num_dimensions);
        clusters[i].R = (float*) malloc(sizeof(float)*num_dimensions*num_dimensions);
        clusters[i].Rinv = (float*) malloc(sizeof(float)*num_dimensions*num_dimensions);
        clusters[i].constant = 0.0;
        clusters[i].p = (float*) malloc(sizeof(float)*num_events);
        clusters[i].w = (float*) malloc(sizeof(float)*num_events);
    }
    
    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));
    
    // Setup the cluster data structures on device
    // First allocate structures on the host, CUDA malloc the arrays
    // Then CUDA malloc structures on the device and copy them over
    cluster* temp_clusters = (cluster*) malloc(sizeof(cluster)*num_clusters);
    for(int i=0; i<num_clusters;i++) {
        temp_clusters[i].N = 0.0;
        temp_clusters[i].pi = 0.0;
        temp_clusters[i].constant = 0.0;
        CUDA_SAFE_CALL(hipMalloc((void**) &(temp_clusters[i].means),sizeof(float)*num_dimensions));
        if(!temp_clusters[i].means) printf("ERROR: Could not allocate memory.\n");
        CUDA_SAFE_CALL(hipMalloc((void**) &(temp_clusters[i].R),sizeof(float)*num_dimensions*num_dimensions));
        if(!temp_clusters[i].R) printf("ERROR: Could not allocate memory.\n");
        CUDA_SAFE_CALL(hipMalloc((void**) &(temp_clusters[i].Rinv),sizeof(float)*num_dimensions*num_dimensions));
        if(!temp_clusters[i].Rinv) printf("ERROR: Could not allocate memory.\n");
        CUDA_SAFE_CALL(hipMalloc((void**) &(temp_clusters[i].p),sizeof(float)*num_events));
        if(!temp_clusters[i].p) printf("ERROR: Could not allocate memory.\n");
        CUDA_SAFE_CALL(hipMalloc((void**) &(temp_clusters[i].w),sizeof(float)*num_events));
        if(!temp_clusters[i].w) printf("ERROR: Could not allocate memory.\n");
    }
    cluster* d_clusters;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_clusters, sizeof(cluster)*num_clusters));
    
    unsigned int mem_size = num_dimensions*num_events*sizeof(float);
    
    // allocate device memory for FCS data
    float* d_fcs_data;
    CUDA_SAFE_CALL(hipMalloc( (void**) &d_fcs_data, mem_size));
    // copy FCS to device
    CUDA_SAFE_CALL(hipMemcpy( d_fcs_data, fcs_data, mem_size,hipMemcpyHostToDevice) );

    // Copy Cluster data to device
    CUDA_SAFE_CALL(hipMemcpy(d_clusters,temp_clusters,sizeof(cluster)*num_clusters,hipMemcpyHostToDevice));
    
    printf("Invoking seed_clusters kernel\n");
    // execute the kernel
    seed_clusters<<< 1, num_threads >>>( d_fcs_data, d_clusters, num_dimensions, num_clusters, num_events);
    
    double determinant;
        
    // Compute new constants and invert matrix
    // copy clusters from the device
    CUDA_SAFE_CALL(hipMemcpy(temp_clusters, d_clusters, sizeof(cluster)*num_clusters,hipMemcpyDeviceToHost));
    for(int i=0; i<num_clusters; i++) {
        // copy the R matrix from the device
        CUDA_SAFE_CALL(hipMemcpy(temp_clusters[i].R, d_clusters[i].R, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyDeviceToHost));

        // copy R into Rinv
        memcpy(temp_clusters[i].Rinv,temp_clusters[i].R,sizeof(float)*num_dimensions*num_dimensions);
        
        // invert the matrix
        //invert_matrix(temp_clusters[i].Rinv,num_dimensions,&determinant);
        invert(temp_clusters[i].Rinv,num_dimensions,&determinant);
        
        // compute the new constant
        temp_clusters[i].constant = (-num_dimensions)*0.5*log(2*3.14159)-0.5*log(fabs(determinant));
        printf("Determinant: %E, new constant: %f\n",fabs(determinant),temp_clusters[i].constant);
        
        // copy the R matrix back to the device
        CUDA_SAFE_CALL(hipMemcpy(d_clusters[i].Rinv, temp_clusters[i].Rinv, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyHostToDevice));
        // copy the new constant to the device
        CUDA_SAFE_CALL(hipMemcpy(&(d_clusters[i].constant), &(temp_clusters[i].constant), sizeof(float),hipMemcpyHostToDevice));            
    }
    
    // Calculate an epsilon value
    int ndata_points = num_events*num_dimensions;
    float epsilon = (1+num_dimensions+0.5*(num_dimensions+1)*num_dimensions)*log((float)ndata_points)*0.01;
    float likelihood, old_likelihood;
    
    epsilon = epsilon*1;
    printf("Gaussian.cu: epsilon = %f\n",epsilon);
    
    float* d_likelihood;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_likelihood, sizeof(float)));

    // do initial regrouping
    printf("Invoking regroup kernel\n");
    regroup<<<1, num_threads>>>(d_fcs_data,d_clusters,num_dimensions,num_clusters,num_events,d_likelihood);
    CUDA_SAFE_CALL(hipMemcpy(&likelihood,d_likelihood,sizeof(float),hipMemcpyDeviceToHost));
    printf("Gaussian.cu: likelihood = %f\n",likelihood);

    float change = epsilon*2;
    
    while(change > epsilon) {
        old_likelihood = likelihood;
        printf("Invoking reestimate_parameters kernel\n");
        reestimate_parameters<<<1, num_threads>>>(d_fcs_data,d_clusters,num_dimensions,num_clusters,num_events);
        
        // Compute new constants and invert matrix
        // copy clusters from the device
        CUDA_SAFE_CALL(hipMemcpy(temp_clusters, d_clusters, sizeof(cluster)*num_clusters,hipMemcpyDeviceToHost));
        for(int i=0; i<num_clusters; i++) {
            // copy the R matrix from the device
            CUDA_SAFE_CALL(hipMemcpy(temp_clusters[i].R, d_clusters[i].R, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyDeviceToHost));

            // copy R into Rinv
            memcpy(temp_clusters[i].Rinv,temp_clusters[i].R,sizeof(float)*num_dimensions*num_dimensions);
            
            // invert the matrix
            //invert_matrix(temp_clusters[i].Rinv,num_dimensions,&determinant);
            invert(temp_clusters[i].Rinv,num_dimensions,&determinant);
            
            // compute the new constant
            temp_clusters[i].constant = (-num_dimensions)*0.5*log(2*3.14159)-0.5*log(fabs(determinant));
            printf("Determinant: %E, new constant: %f\n",fabs(determinant),temp_clusters[i].constant);
            
            // copy the R matrix back to the device
            CUDA_SAFE_CALL(hipMemcpy(d_clusters[i].Rinv, temp_clusters[i].Rinv, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyHostToDevice));
            // copy the new constant to the device
            CUDA_SAFE_CALL(hipMemcpy(&(d_clusters[i].constant), &(temp_clusters[i].constant), sizeof(float),hipMemcpyHostToDevice));            
        }
        
        printf("Invoking regroup kernel\n");
        regroup<<<1, num_threads>>>(d_fcs_data,d_clusters,num_dimensions,num_clusters,num_events,d_likelihood);
        CUDA_SAFE_CALL(hipMemcpy(&likelihood,d_likelihood,sizeof(float),hipMemcpyDeviceToHost));
        printf("likelihood = %f\n",likelihood);
        change = likelihood - old_likelihood;
        printf("Change in likelihood: %f\n",change);
    }

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
    
    // copy clusters from the device
    CUDA_SAFE_CALL(hipMemcpy(temp_clusters, d_clusters, sizeof(cluster)*num_clusters,hipMemcpyDeviceToHost));
    // copy all of the arrays from the structs
    for(int i=0; i<num_clusters; i++) {
        CUDA_SAFE_CALL(hipMemcpy(clusters[i].means, temp_clusters[i].means, sizeof(float)*num_dimensions,hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(clusters[i].R, temp_clusters[i].R, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(clusters[i].Rinv, temp_clusters[i].Rinv, sizeof(float)*num_dimensions*num_dimensions,hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(clusters[i].p, temp_clusters[i].p, sizeof(float)*num_events,hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(clusters[i].w, temp_clusters[i].w, sizeof(float)*num_events,hipMemcpyDeviceToHost));
        clusters[i].N = temp_clusters[i].N;
        clusters[i].pi = temp_clusters[i].pi;
        clusters[i].constant = temp_clusters[i].constant;
    }
    
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    
    for(int c=0; c<num_clusters; c++) {
        printCluster(clusters[c],num_dimensions);
    }
    // cleanup memory
    free(fcs_data);
    for(int i=0; i<num_clusters; i++) {
        free(clusters[i].means);
        free(clusters[i].R);
        free(clusters[i].Rinv);
        free(clusters[i].p);
        free(clusters[i].w);
    }    
    free(clusters);
    CUDA_SAFE_CALL(hipFree(d_fcs_data));
    for(int i=0; i<num_clusters; i++) {
        CUDA_SAFE_CALL(hipFree(temp_clusters[i].means));
        CUDA_SAFE_CALL(hipFree(temp_clusters[i].R));
        CUDA_SAFE_CALL(hipFree(temp_clusters[i].Rinv));
        CUDA_SAFE_CALL(hipFree(temp_clusters[i].p));
        CUDA_SAFE_CALL(hipFree(temp_clusters[i].w));
    }
    free(temp_clusters);
    CUDA_SAFE_CALL(hipFree(d_clusters));

    return 0;
}
