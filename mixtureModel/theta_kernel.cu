#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code.
 */

#define COVARIANCE_DYNAMIC_RANGE 1E5

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "gaussian.h"

#define sdata(index)      CUT_BANK_CHECKER(sdata, index)

/*
 * Compute the spectral means of the FCS data
 */ 
__device__ void spectralMean(float* fcs_data, int num_dimensions, int num_events, float* means) {
    // access thread id
    int tid = threadIdx.x;
    // access number of threads in this block
    int num_threads = blockDim.x;

    if(tid < num_dimensions) {
        means[tid] = 0.0;
    }

    __syncthreads();

    int num_data_points = num_events*num_dimensions;

    // Sum up all the values for the dimension
    for(int i=tid; i < num_data_points; i+= num_dimensions) {
        if(tid < num_dimensions) {
            means[tid] += fcs_data[i];
        }  
    }
    
    __syncthreads();

    // Divide by the # of elements to get the average
    if(tid < num_dimensions) {
        means[tid] /= (float) num_events;
    }
}

__device__ void averageVariance(float* fcs_data, float* means, int num_dimensions, int num_events, float* avgvar) {
    // access thread id
    int tid = threadIdx.x;
    // access number of threads
    int num_threads = blockDim.x;
    
    __shared__ float variances[NUM_DIMENSIONS];
    __shared__ float total_variance;
    
    // Compute average variance for each dimension
    for(int i=0; i < num_dimensions; i += num_threads) {
        if(tid+i < num_dimensions) {
            variances[tid] = 0.0;
            // Sum up all the variance
            for(int j=0; j < num_events; j++) {
                // variance = (data - mean)^2
                //variances[tid+i] += (fcs_data[j*num_dimensions + tid + i]-means[tid+i])*(fcs_data[j*num_dimensions + tid + i]-means[tid+i]);
                variances[tid+i] += (fcs_data[j*num_dimensions + tid + i])*(fcs_data[j*num_dimensions + tid + i]);
            }
            variances[tid+i] /= (float) num_events;
            variances[tid+i] -= means[tid+i]*means[tid+i];
        }
    }
    
    __syncthreads();
    
    if(tid == 0) {
        total_variance = 0.0;
        for(int i=0; i<num_dimensions;i++) {
            ////printf("%f ",variances[tid]);
            total_variance += variances[i];
        }
        ////printf("\nTotal variance: %f\n",total_variance);
        *avgvar = total_variance / (float) num_dimensions;
        ////printf("Average Variance: %f\n",*avgvar);
    }
}

__device__ void invert(float* data, int actualsize, float* determinant)  {
    int maxsize = actualsize;
    int n = actualsize;
    
    if(threadIdx.x == 0) {
    *determinant = 1.0;

#if EMU
    if(VERBOSE) {
        printf("\n\nR matrix before inversion:\n");
        for(int i=0; i<n; i++) {
            for(int j=0; j<n; j++) {
                printf("%.2f ",data[i*n+j]);
            }
            printf("\n");
        }
    }
#endif
    
  if (actualsize <= 0) return;  // sanity check
  if (actualsize == 1) return;  // must be of dimension >= 2
  for (int i=1; i < actualsize; i++) data[i] /= data[0]; // normalize row 0
  for (int i=1; i < actualsize; i++)  { 
    for (int j=i; j < actualsize; j++)  { // do a column of L
      float sum = 0.0;
      for (int k = 0; k < i; k++)  
          sum += data[j*maxsize+k] * data[k*maxsize+i];
      data[j*maxsize+i] -= sum;
      }
    if (i == actualsize-1) continue;
    for (int j=i+1; j < actualsize; j++)  {  // do a row of U
      float sum = 0.0;
      for (int k = 0; k < i; k++)
          sum += data[i*maxsize+k]*data[k*maxsize+j];
      data[i*maxsize+j] = 
         (data[i*maxsize+j]-sum) / data[i*maxsize+i];
      }
    }
    
    for(int i=0; i<actualsize; i++) {
        *determinant *= data[i*n+i];
    }
#if EMU
    if(VERBOSE) {
        printf("Determinant: %E\n",*determinant);
    }
#endif
    
  for ( int i = 0; i < actualsize; i++ )  // invert L
    for ( int j = i; j < actualsize; j++ )  {
      float x = 1.0;
      if ( i != j ) {
        x = 0.0;
        for ( int k = i; k < j; k++ ) 
            x -= data[j*maxsize+k]*data[k*maxsize+i];
        }
      data[j*maxsize+i] = x / data[j*maxsize+j];
      }
  for ( int i = 0; i < actualsize; i++ )   // invert U
    for ( int j = i; j < actualsize; j++ )  {
      if ( i == j ) continue;
      float sum = 0.0;
      for ( int k = i; k < j; k++ )
          sum += data[k*maxsize+j]*( (i==k) ? 1.0 : data[i*maxsize+k] );
      data[i*maxsize+j] = -sum;
      }
  for ( int i = 0; i < actualsize; i++ )   // final inversion
    for ( int j = 0; j < actualsize; j++ )  {
      float sum = 0.0;
      for ( int k = ((i>j)?i:j); k < actualsize; k++ )  
          sum += ((j==k)?1.0:data[j*maxsize+k])*data[k*maxsize+i];
      data[j*maxsize+i] = sum;
      }
      
#if EMU
      if(VERBOSE) {
          printf("\n\nR matrix after inversion:\n");
          for(int i=0; i<n; i++) {
              for(int j=0; j<n; j++) {
                  printf("%.2f ",data[i*n+j]);
              }
              printf("\n");
          }
      }
#endif
    }
 }


__device__ void normalize_pi(cluster* clusters, int num_clusters) {
    __shared__ float sum;
    
    // TODO: could maybe use a parallel reduction..but the # of elements is really small
    // What is better: having thread 0 compute a shared sum and sync, or just have each one compute the sum?
    if(threadIdx.x == 0) {
        sum = 0.0;
        for(int i=0; i<num_clusters; i++) {
            sum += clusters[i].pi;
        }
    }
    
    __syncthreads();
    
    if(threadIdx.x < num_clusters) {
        if(sum > 0.0) {
            clusters[threadIdx.x].pi /= sum;
        } else {
            clusters[threadIdx.x].pi = 0.0;
        }
    }
    
    __syncthreads();
}


__device__ void compute_constants(cluster* clusters, int num_clusters, int num_dimensions) {
    const int tid = threadIdx.x;
    const int num_threads = blockDim.x;
    const int num_elements = num_dimensions*num_dimensions;
    
    __shared__ float determinant_arg;
    
    float determinant;
    
    __shared__ float matrix[NUM_DIMENSIONS*NUM_DIMENSIONS];
    
    // Invert the matrix for every cluster
    for(int c=blockIdx.x; c < num_clusters; c+=NUM_BLOCKS) {
        // Copy the R matrix into shared memory for doing the matrix inversion
        for(int i=tid; i<num_elements; i+= num_threads ) {
            matrix[i] = clusters[c].R[i];
        }
        
        __syncthreads(); 

        invert(matrix,num_dimensions,&determinant_arg);

        __syncthreads(); 
        
        determinant = determinant_arg;
        
        // Copy the matrx from shared memory back into the cluster memory
        for(int i=tid; i<num_elements; i+= num_threads) {
            clusters[c].Rinv[i] = matrix[i];
        }
        
        __syncthreads();
        
        // Compute the constant
        if(tid == 0) {
            determinant = fabs(determinant);
            clusters[c].constant = -num_dimensions*0.5*logf(2*PI) - 0.5*logf(determinant);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata          FCS data: [num_events]
//! @param clusters         Clusters: [num_clusters]
//! @param num_dimensions   number of dimensions in an FCS event
//! @param num_events       number of FCS events
////////////////////////////////////////////////////////////////////////////////
__global__ void
seed_clusters( float* g_idata, cluster* clusters, int num_dimensions, int num_clusters, int num_events) 
{
    // access thread id
    int tid = threadIdx.x;
    // access number of threads in this block
    int num_threads = blockDim.x;

    // shared memory
    __shared__ float means[NUM_DIMENSIONS];
    
    // Compute the means
    spectralMean(g_idata, num_dimensions, num_events, means);

    __syncthreads();
    
    float avgvar;
    
    // Compute the average variance
    averageVariance(g_idata, means, num_dimensions, num_events, &avgvar);
        
    // Initialize covariances
    __shared__ float covs[NUM_DIMENSIONS*NUM_DIMENSIONS]; 
    
    int num_elements;
    int row, col;
        
    // Number of elements in the covariance matrix
    num_elements = num_dimensions*num_dimensions; 

    __syncthreads();

    // Compute the initial covariance matrix of the data
    for(int i=tid; i < num_elements; i+= num_threads) {
            // zero the value, find what row and col this thread is computing
            covs[i] = 0.0;
            row = (i) / num_dimensions;
            col = (i) % num_dimensions;

            for(int j=0; j < num_events; j++) {
                covs[i] += (g_idata[j*num_dimensions+row])*(g_idata[j*num_dimensions+col]); 
            }
            covs[i] = covs[i] / (float) num_events;
            covs[i] = covs[i] - means[row]*means[col];
    } 
    __syncthreads();    
    
    // Calculate a seed value for the means
    float seed;
    if(num_clusters > 1) {
        seed = (num_events-1.0)/(num_clusters-1.0);
    } else {
        seed = 0.0;
    }
    
    __syncthreads();
    
    // Seed the pi, means, and covariances for every cluster
    for(int c=0; c < num_clusters; c++) {
        clusters[c].pi = 1.0/num_clusters;
        if(tid < num_dimensions) {
            clusters[c].means[tid] = g_idata[((int)(c*seed))*num_dimensions+tid];
            //clusters[c].means[tid] = means[tid];
        }
          
        for(int i=tid; i < num_elements; i+= num_threads) {
            // Add the average variance divided by a constant, this keeps the cov matrix from becoming singular
            clusters[c].R[i] = covs[i] + avgvar/COVARIANCE_DYNAMIC_RANGE;
        }
        
        clusters[c].avgvar = avgvar / COVARIANCE_DYNAMIC_RANGE;
    }
}

__global__ void
regroup(float* fcs_data, cluster* clusters, int num_dimensions, int num_clusters, int num_events, float* likelihood) {
    float like;
    float max_likelihood;
    float denominator_sum;
    float temp;
    float thread_likelihood = 0.0;
    __shared__ float total_likelihoods[NUM_THREADS];
    
    const int num_threads = blockDim.x;
    int num_pixels_per_block = num_events / NUM_BLOCKS;  
    const int tid = threadIdx.x;
    
    int start_index;
    int end_index;
    start_index = blockIdx.x * num_pixels_per_block + tid;
    
    if(blockIdx.x == NUM_BLOCKS-1) {
        end_index = num_events;
    } else {
        end_index = (blockIdx.x+1) * num_pixels_per_block;
    }
    
    //printf("Block Index: %d, Thread Index: %d, start_index: %d, end_index: %d\n",blockIdx.x,tid,start_index,end_index);

    int data_index;
    
    total_likelihoods[tid] = 0.0;
  
#if EMU
    if(VERBOSE) { 
        for(int c=0;c<num_clusters;c++) {
            if(tid==0) {
                printf("cluster[%d].Rinv matrix:\n",c);
                for(int i=0;i<num_dimensions;i++) {
                    for(int j=0; j<num_dimensions;j++) {
                        printf("%.3f ",clusters->Rinv[i*num_dimensions+j]);
                    }
                    printf("\n");
                }
            }
        }
    }
#endif
    
    
    // Compute likelihood for every event, for every cluster
    for(int pixel=start_index; pixel<end_index; pixel += num_threads) {
       
        data_index = pixel*num_dimensions;
        // compute likelihood of pixel in cluster 'c'
        for(int c=0; c<num_clusters; c++) {
            like = 0.0;
            // this does the loglike() function
            for(int i=0; i<num_dimensions; i++) {
                for(int j=0; j<num_dimensions; j++) {
                    like += (fcs_data[data_index+i]-clusters[c].means[i])*(fcs_data[data_index+j]-clusters[c].means[j])*clusters[c].Rinv[i*num_dimensions+j];
                }
            }
            temp = -0.5*like+clusters[c].constant;
            clusters[c].p[pixel] = temp;
 
            // Keep track of the maximum likelihood
            if(c == 0) {
                max_likelihood = temp;
            } 
            if( temp > max_likelihood) {
                max_likelihood = temp;
            }
        }
        
        denominator_sum = 0.0;
        for(int c=0; c<num_clusters; c++) {
            temp = exp(clusters[c].p[pixel]-max_likelihood)*clusters[c].pi;
            denominator_sum += temp;
            clusters[c].p[pixel] = temp;
        }
        thread_likelihood += log(denominator_sum) + max_likelihood;
        
        // Normalizes probabilities
        for(int c=0; c<num_clusters; c++) {
            clusters[c].p[pixel] /= denominator_sum;
            //printf("Probability that pixel #%d is in cluster #%d: %f\n",pixel,c,clusters[c].p[pixel]);
        }
    }
    
    total_likelihoods[tid] = thread_likelihood;

    
    float retval = 0.0;
    
    __syncthreads();
    
    // Reduce all the total_likelihoods to a single total
    if(tid == 0) {
        for(int i=0; i<num_threads; i++) {
            retval += total_likelihoods[i];
        }
        likelihood[blockIdx.x] = retval;
    }
}

/*
 * This kernel re-computes the means, N (number of data points per cluster),
 * and R (covariance matrix). The computations for each cluster are independent
 * therefore each cluster can be computed by a different block
 */
__global__ void
reestimate_parameters(float* fcs_data, cluster* clusters, int num_dimensions, int num_clusters, int num_events) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    
    // Number of elements in the covariance matrix
    int num_elements = num_dimensions*num_dimensions;
    
    // Figure out # of elements each thread should add up
    int num_elements_per_thread = num_events / num_threads;
    int start_index = tid * num_elements_per_thread;
    int end_index;
    // handle the end block so that we add left-over elements too
    if(threadIdx.x == (num_threads-1)) {
        end_index = num_events;
    } else {
        end_index = start_index + num_elements_per_thread;
    }

    // Need to store the sum computed by each thread so in the end
    // a single thread can reduce to get the final sum
    __shared__ float temp_sums[NUM_THREADS];

    // Store the means in shared memory to speed up the covariance computations
    __shared__ float means[NUM_DIMENSIONS];
 
    // Compute new N
    for(int c=blockIdx.x; c<num_clusters; c += NUM_BLOCKS) {
        temp_sums[tid] = 0.0;
        // Break all the events accross the threads, add up probabilities
        for(int s=start_index; s<end_index; s++) {
            temp_sums[tid] += clusters[c].p[s];
        }
        
        __syncthreads();
        
        // Let the first thread add up all the intermediate sums
        if(tid == 0) {
            clusters[c].N = 0.0;
            for(int j=0; j<num_threads; j++) {
                clusters[c].N += temp_sums[j];
            }
            //printf("clusters[%d].N = %f\n",c,clusters[c].N);
            
            // Set PI to the # of expected items, and then normalize it later
            clusters[c].pi = clusters[c].N;
        }
    }

    // Synchronize because threads need to use clusters[c].N for means calculation    
    __syncthreads();

    float mean_sum;   
    float cov_sum = 0.0;
    int row,col,data_index;
   
    cluster* clust;
    // Compute means and covariances for each subcluster
    for(int c=blockIdx.x; c<num_clusters; c += NUM_BLOCKS) {
        clust = &(clusters[c]);
        
        // Compute means
        //  Let one thread handle each dimension
        //  There are only 8 cores per multiprocessor so I don't think we're really wasting
        //  resources badly by doing it this way. It's got alot fewer loops and potential branching
        //  than doing it like the N computation above
        if(tid < num_dimensions) {    
            mean_sum = 0.0;
            for(int s=0; s<num_events; s++) {
                mean_sum += fcs_data[s*num_dimensions+tid]*clust->p[s];
            }
            // Divide by # of elements in the cluster
            means[tid] = mean_sum / clust->N;
            clust->means[tid] = means[tid];
        }

        __syncthreads();

        // Compute the covariance matrix of the data
        for(int i=tid; i < num_elements; i+= num_threads) {
            // zero the value, find what row and col this thread is computing
            cov_sum = 0.0;
            row = (i) / num_dimensions;
            col = (i) % num_dimensions;
            data_index = 0;

            for(int j=0; j < num_events; j++) {
                cov_sum += (fcs_data[data_index+row]-means[row])*(fcs_data[data_index+col]-means[col])*clust->p[j]; 
                data_index += num_dimensions;
            }
            clust->R[i] = cov_sum / clust->N;
        }
        
        __syncthreads();

        // Regularize matrix
        if(tid < num_dimensions) {
            clust->R[tid*num_dimensions+tid] += clust->avgvar;
        }
    }
}

/*
 * Computes the constant for each cluster and normalizes pi for every cluster
 * In the process it inverts R and finds the determinant
 */
__global__ void
constants_kernel(cluster* clusters, int num_clusters, int num_dimensions) {
    compute_constants(clusters,num_clusters,num_dimensions);
    
    __syncthreads();
    
    if(blockIdx.x == 0) {
        normalize_pi(clusters,num_clusters);
    }
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
