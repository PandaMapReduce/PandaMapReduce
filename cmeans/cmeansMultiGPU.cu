#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cmeansMultiGPU.h>
#include <cmeansMultiGPUcu.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
//#include <cmeans_kernel.cu>
#include "MDL.h"

void printCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }
}

typedef struct {
    hipEvent_t start;
    hipEvent_t stop;
    float* et;
} cudaTimer_t;

void createTimer(cudaTimer_t* timer) {
    #pragma omp critical (create_timer) 
    {
        hipEventCreate(&(timer->start));
        hipEventCreate(&(timer->stop));
        timer->et = (float*) malloc(sizeof(float));
        *(timer->et) = 0.0f;
    }
}

void deleteTimer(cudaTimer_t timer) {
    #pragma omp critical (delete_timer) 
    {
        hipEventDestroy(timer.start);
        hipEventDestroy(timer.stop);
        free(timer.et);
    }
}

void startTimer(cudaTimer_t timer) {
    hipEventRecord(timer.start,0);
}

void stopTimer(cudaTimer_t timer) {
    hipEventRecord(timer.stop,0);
    hipEventSynchronize(timer.stop);
    float tmp;
    hipEventElapsedTime(&tmp,timer.start,timer.stop);
    *(timer.et) += tmp;
}

float getTimerValue(cudaTimer_t timer) {
    return *(timer.et);
}

/************************************************************************/
/* C-means Main                                                            */
/************************************************************************/
int main(int argc, char* argv[])
{
    cudaTimer_t timer_io; // Timer for I/O, such as reading FCS file and outputting result files
    cudaTimer_t timer_total; // Total time
   
    createTimer(&timer_io);
    createTimer(&timer_total);
    
    startTimer(timer_total);
    startTimer(timer_io);
    
    // [program name]  [data file]
    if(argc != 2){
        printf("Usage Error: must supply data file. e.g. programe_name @opt(flags) file.in\n");
        return 1;
    }

    float* myEvents = ParseSampleInput(argv[1]);
#if FAKE
    free(myEvents);
    myEvents = generateEvents();
#endif
    if(myEvents == NULL){
        return 1;
    }
     
    printf("Parsed file\n");
    
    int num_gpus = 0;       // number of CUDA GPUs

    // determine the number of CUDA capable GPUs
    hipGetDeviceCount(&num_gpus);
    if(num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }

    // display CPU and GPU configuration
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);
    for(int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }
    printf("---------------------------\n");
    
    //srand((unsigned)(time(0)));
    srand(42);
    
    stopTimer(timer_io);
    
    // Allocate arrays for the cluster centers
    float* myClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    float* newClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);

    // Select random cluster centers
    generateInitialClusters(myClusters, myEvents);

    // Create an array of arrays for temporary cluster centers from each GPU
    float** tempClusters = (float**) malloc(sizeof(float*)*num_gpus);
    float** tempDenominators = (float**) malloc(sizeof(float*)*num_gpus);
    for(int i=0; i < num_gpus; i++) {
        tempClusters[i] = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
        tempDenominators[i] = (float*) malloc(sizeof(float)*NUM_CLUSTERS);
        memcpy(tempClusters[i],myClusters,sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    }
    // Create an array of arrays for temporary Q matrix pieces from each GPU
    float** q_matrices = (float**) malloc(sizeof(float*)*num_gpus);
    // Create an array for the final Q matrix
    float* q_matrix = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_CLUSTERS);
    
    float diff; // used to track difference in cluster centers between iterations

    // Transpose the events matrix
    float* transposedEvents = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    for(int i=0; i<NUM_EVENTS; i++) {
        for(int j=0; j<NUM_DIMENSIONS; j++) {
            transposedEvents[j*NUM_EVENTS+i] = myEvents[i*NUM_DIMENSIONS+j];
        }
    }
   
    ////////////////////////////////////////////////////////////////
    // run as many CPU threads as there are CUDA devices
    //   each CPU thread controls a different device, processing its
    //   portion of the data.  It's possible to use more CPU threads
    //   than there are CUDA devices, in which case several CPU
    //   threads will be allocating resources and launching kernels
    //   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
    //   Recall that all variables declared inside an "omp parallel" scope are
    //   local to each CPU thread
    //
    //num_gpus = 1;
    omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    //omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there are CUDA devices
    #pragma omp parallel shared(myClusters,diff,tempClusters,tempDenominators)
    {
        cudaTimer_t timer_memcpy; // Timer for GPU <---> CPU memory copying
        cudaTimer_t timer_cpu; // Timer for processing on CPU
        cudaTimer_t timer_gpu; // Timer for kernels on the GPU
        
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        printf("hello from thread %d of %d\n",cpu_thread_id,num_cpu_threads);

        // set and check the CUDA device for this CPU thread
        int gpu_id = -1;
        hipSetDevice(cpu_thread_id % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
        hipGetDevice(&gpu_id);
       
        #pragma omp barrier
 
        createTimer(&timer_memcpy);
        createTimer(&timer_cpu);
        createTimer(&timer_gpu);

        printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);
#if !CPU_ONLY    
        startTimer(timer_memcpy);
        float* d_distanceMatrix;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_distanceMatrix, sizeof(float)*NUM_EVENTS*NUM_CLUSTERS));
        float* d_E;// = AllocateEvents(myEvents);
        CUDA_SAFE_CALL(hipMalloc((void**)&d_E, sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS));
        float* d_C;// = AllocateClusters(myClusters);
        CUDA_SAFE_CALL(hipMalloc((void**)&d_C, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
        float* d_nC;// = AllocateCM(cM);
        CUDA_SAFE_CALL(hipMalloc((void**)&d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
        float* d_denoms;// = AllocateCM(cM);
        CUDA_SAFE_CALL(hipMalloc((void**)&d_denoms, sizeof(float)*NUM_CLUSTERS));
        int size = sizeof(float)*NUM_DIMENSIONS*NUM_EVENTS;
        //CUDA_SAFE_CALL(hipMemcpy(d_E, myEvents, size, hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(d_E, transposedEvents, size, hipMemcpyHostToDevice));
        size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;
        CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
        stopTimer(timer_memcpy);
#endif
        clock_t cpu_start, cpu_stop;
        cpu_start = clock();
        printf("Starting C-means\n");
        float averageTime = 0.0f;
        int iterations = 0;
        
        // Compute starting/finishing indexes for the events for each gpu
        int start = cpu_thread_id*NUM_EVENTS/num_gpus;
        int finish = (cpu_thread_id+1)*NUM_EVENTS/num_gpus;
        if(cpu_thread_id == (num_gpus-1)) {
            finish = NUM_EVENTS;
        }
        printf("GPU %d, Starting Event: %d, Ending Event: %d\n",cpu_thread_id,start,finish);

        do{
#if CPU_ONLY
            startTimer(timer_cpu);
            if (cpu_thread_id == 0) {
                clock_t cpu_start, cpu_stop;
                cpu_start = clock();

                UpdateClusterCentersCPU(myClusters, myEvents, newClusters);

                cpu_stop = clock();
                printf("Processing time for CPU: %f (ms) \n", (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC)*(float)1e3);
                averageTime += (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC)*(float)1e3;
                
                for(int i=0; i < NUM_CLUSTERS; i++){
                    for(int k = 0; k < NUM_DIMENSIONS; k++){
                        diff += fabs(newClusters[i*NUM_DIMENSIONS + k] - myClusters[i*NUM_DIMENSIONS + k]);
                    }
                }
                memcpy(myClusters,newClusters,sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS);
            }
            stopTimer(timer_cpu);
#else
            cudaTimer_t timer;
            createTimer(&timer);
            startTimer(timer);

            size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;

            // Copy the cluster centers to the GPU
            startTimer(timer_memcpy);
            CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
            stopTimer(timer_memcpy);
            
            dim3 BLOCK_DIM(1, NUM_THREADS, 1);

            startTimer(timer_gpu);
            printf("Launching ComputeDistanceMatrix kernel\n");
            ComputeDistanceMatrix<<< NUM_CLUSTERS, 320  >>>(d_C, d_E, d_distanceMatrix, start, finish);
            hipDeviceSynchronize();
            printCudaError();
            printf("Launching UpdateClusterCentersGPU kernel\n");
            UpdateClusterCentersGPU<<< NUM_BLOCKS, NUM_THREADS >>>(d_C, d_E, d_nC, d_distanceMatrix, d_denoms, start, finish);
            hipDeviceSynchronize();
            printCudaError();
            
            stopTimer(timer_gpu);
            
            // Copy partial centers and denominators to host
            startTimer(timer_memcpy);
            hipMemcpy(tempClusters[cpu_thread_id], d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS, hipMemcpyDeviceToHost);
            hipMemcpy(tempDenominators[cpu_thread_id], d_denoms, sizeof(float)*NUM_CLUSTERS, hipMemcpyDeviceToHost);
            printCudaError();
            stopTimer(timer_memcpy);
            
            stopTimer(timer);
            float thisTime = getTimerValue(timer);
            printf("Processing time for GPU %d: %f (ms) \n", cpu_thread_id, thisTime);
            averageTime += thisTime;
            deleteTimer(timer);

            startTimer(timer_cpu);
        
            #pragma omp barrier
            if(cpu_thread_id == 0) {
                // Sum up the partial cluster centers (numerators)
                for(int i=1; i < num_gpus; i++) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        for(int d=0; d < NUM_DIMENSIONS; d++) {
                            tempClusters[0][c*NUM_DIMENSIONS+d] += tempClusters[i][c*NUM_DIMENSIONS+d];
                        }
                    }
                }

                // Sum up the denominator for each cluster
                for(int i=1; i < num_gpus; i++) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        tempDenominators[0][c] += tempDenominators[i][c];
                    }
                }

                // Divide to get the final clusters
                for(int c=0; c < NUM_CLUSTERS; c++) {
                    for(int d=0; d < NUM_DIMENSIONS; d++) {
                        tempClusters[0][c*NUM_DIMENSIONS+d] /= tempDenominators[0][c];
                    }
                }
                diff = 0.0;
                for(int i=0; i < NUM_CLUSTERS; i++){
                    //printf("GPU %d, Cluster %d: ",cpu_thread_id,i);
                    for(int k = 0; k < NUM_DIMENSIONS; k++){
                        //printf("%f ",tempClusters[cpu_thread_id][i*NUM_DIMENSIONS + k]);
                        diff += fabs(myClusters[i*NUM_DIMENSIONS + k] - tempClusters[cpu_thread_id][i*NUM_DIMENSIONS + k]);
                    }
                    //printf("\n");
                }
                memcpy(myClusters,tempClusters[cpu_thread_id],sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS);
                printf("Diff = %f\n", diff);
                printf("Done with iteration #%d\n", iterations);
                fflush(stdout);
            }
            stopTimer(timer_cpu);
            
#endif
            #pragma omp barrier
            iterations++;
            printf("\n");
        } while(abs(diff) > THRESHOLD && iterations < 150); 

        if(cpu_thread_id == 0) {        
            if(iterations == 150){
                printf("Warning: c-means did not converge to the %f threshold provided\n", THRESHOLD);
            }
            cpu_stop = clock();
            
            startTimer(timer_io);
            
            averageTime /= iterations;
            printf("\nTotal Processing time: %f (s) \n", (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC));
            printf("C-means complete\n");
            printf("\n");

            for(int i=0; i < NUM_CLUSTERS; i++){
                printf("GPU %d, Center %d: ",cpu_thread_id,i);
                for(int k = 0; k < NUM_DIMENSIONS; k++)
                    printf("%f\t", myClusters[i*NUM_DIMENSIONS + k]);
                printf("\n");
            }
            
            stopTimer(timer_io);
        }
        
        #pragma omp barrier // sync threads 
            
        int* finalClusterConfig;
        float mdlTime = 0;
        
        #if !MDL_on_GPU
            finalClusterConfig = MDL(myEvents, myClusters, &mdlTime, argv[1]);
        #else
            printf("Calculating Q Matrix Section %d\n",cpu_thread_id);
           
            // Copy the latest clusters to the device 
            //  (the current ones on the device are 1 iteration old) 
            startTimer(timer_memcpy);
            CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
            stopTimer(timer_memcpy);
            
            // Build Q matrix, each gpu handles NUM_DIMENSIONS/num_gpus rows of the matrix
            q_matrices[cpu_thread_id] = BuildQGPU(d_E, d_C, d_distanceMatrix, &mdlTime, cpu_thread_id, num_gpus);
            
            #pragma omp barrier // sync threads
            
            if(cpu_thread_id == 0) {
                // Combine the partial matrices
                int num_matrix_elements = NUM_CLUSTERS*(NUM_CLUSTERS/num_gpus);
                for(int i=0; i < num_gpus; i++) {
                    float* q_matrix_ptr = (float*) q_matrix+i*num_matrix_elements;
                    float* q_matrices_ptr = (float*) q_matrices[i]+i*num_matrix_elements;
                    memcpy(q_matrix_ptr,q_matrices_ptr,sizeof(float)*num_matrix_elements);   
                    free(q_matrices[i]);
                }
                startTimer(timer_cpu);
                printf("Searching for optimal configuration...\n");
                finalClusterConfig = TabuSearch(q_matrix, argv[1]);
                stopTimer(timer_cpu);

                printf("Q Matrix:\n");
                for(int row=0; row < NUM_CLUSTERS; row++) {
                    for(int col=0; col < NUM_CLUSTERS; col++) {
                        printf("%.2e ",q_matrix[row*NUM_CLUSTERS+col]);
                    }
                    printf("\n");
                }
                
                free(q_matrix);
            }
            mdlTime /= 1000.0; // CUDA timer returns time in milliseconds, normalize to seconds
        #endif

 
        if(cpu_thread_id == 0) {        
            startTimer(timer_io);

            printf("Final Clusters are:\n");
            int newCount = 0;
            for(int i = 0; i < NUM_CLUSTERS; i++){
                if(finalClusterConfig[i]){
                    for(int j = 0; j < NUM_DIMENSIONS; j++){
                        newClusters[newCount * NUM_DIMENSIONS + j] = myClusters[i*NUM_DIMENSIONS + j];
                        printf("%f\t", myClusters[i*NUM_DIMENSIONS + j]);
                    }
                    newCount++;
                    printf("\n");
                }
            }
            
            fflush(stdout);
            //exit(1);
            FindCharacteristics(myEvents, newClusters, newCount, argv[1]);
            stopTimer(timer_io);
        }
        printf("\n\n"); 
        printf("Thread %d: GPU memcpy Time (ms): %f\n",cpu_thread_id,getTimerValue(timer_memcpy));
        printf("Thread %d: CPU processing Time (ms): %f\n",cpu_thread_id,getTimerValue(timer_cpu));
        printf("Thread %d: GPU processing Time (ms): %f\n",cpu_thread_id,getTimerValue(timer_gpu));
        
        #if !CPU_ONLY
            CUDA_SAFE_CALL(hipFree(d_E));
            CUDA_SAFE_CALL(hipFree(d_C));
            CUDA_SAFE_CALL(hipFree(d_nC));
        #endif
    
        #pragma omp barrier
        printf("Thread %d done.\n",cpu_thread_id);
    } // end of omp_parallel block
    stopTimer(timer_total);
    
    printf("Total Time (ms): %f\n",getTimerValue(timer_total));
    printf("I/O Time (ms): %f\n",getTimerValue(timer_io));
    printf("\n\n"); 
    
    free(newClusters);
    free(myClusters);
    free(myEvents);
    free(transposedEvents);
    return 0;
}

float* generateEvents(){
    float* allEvents = (float*) malloc(NUM_EVENTS*NUM_DIMENSIONS*sizeof(float));
    //generateEvents around (10,10,10), (20, 10, 50), and (50, 50, 0)
    int i, j;
    for(i = 0; i < NUM_EVENTS; i++){
        for(j =0; j < 3; j++){
                
        if(i < NUM_EVENTS/3){
            allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 7;
        }
        else if(i < NUM_EVENTS*2/3){
            switch(j){
                case 0: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 47; break;
                case 1: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 27; break;
                case 2: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 7; break;
                default: printf("error!\n");
            }
        }
        else {
            switch(j){
                case 0: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 47; break;
                case 1: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*6 + 47; break;
                case 2: allEvents[i*3 + j] = rand()/(float(RAND_MAX)+1)*3 ; break;
                default: printf("error!\n");
            }

        }
        }
    }
    return allEvents;
}

void generateInitialClusters(float* clusters, float* events){
    int seed;
    for(int i = 0; i < NUM_CLUSTERS; i++){
        seed = rand() % NUM_EVENTS;
        for(int j = 0; j < NUM_DIMENSIONS; j++){
            clusters[i*NUM_DIMENSIONS + j] = events[seed*NUM_DIMENSIONS + j];
        }
    }
    
}



__host__ float CalculateDistanceCPU(const float* clusters, const float* events, int clusterIndex, int eventIndex){

    float sum = 0;
#if DISTANCE_MEASURE == 0
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i];
        sum += tmp*tmp;
    }
    sum = sqrt(sum);
#endif
#if DISTANCE_MEASURE == 1
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i];
        sum += abs(tmp);
    }
#endif
#if DISTANCE_MEASURE == 2
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = abs(events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i]);
        if(tmp > sum)
            sum = tmp;
    }
#endif
    return sum;
}


__host__ float MembershipValue(const float* clusters, const float* events, int clusterIndex, int eventIndex){
    float myClustDist = CalculateDistanceCPU(clusters, events, clusterIndex, eventIndex);
    float sum =0;
    float otherClustDist;
    for(int j = 0; j< NUM_CLUSTERS; j++){
        otherClustDist = CalculateDistanceCPU(clusters, events, j, eventIndex); 
        if(otherClustDist < .000001)
            return 0.0;
        sum += pow((float)(myClustDist/otherClustDist),float(2/(FUZZINESS-1)));
    }
    return 1/sum;
}



void UpdateClusterCentersCPU(const float* oldClusters, const float* events, float* newClusters){
    
    
    //float membershipValue, sum, denominator;
    float membershipValue, denominator;
    float* numerator = (float*)malloc(sizeof(float)*NUM_DIMENSIONS);
    float* denominators = (float*)malloc(sizeof(float)*NUM_CLUSTERS);
    float* distances = (float*)malloc(sizeof(float)*NUM_CLUSTERS);

    
    for(int i = 0; i < NUM_CLUSTERS; i++){
      denominator = 0.0;
      for(int j = 0; j < NUM_DIMENSIONS; j++)
        numerator[j] = 0;
      for(int j = 0; j < NUM_EVENTS; j++){
        membershipValue = MembershipValue(oldClusters, events, i, j);
        for(int k = 0; k < NUM_DIMENSIONS; k++){
          numerator[k] += events[j*NUM_DIMENSIONS + k]*membershipValue;
        }
        
        denominator += membershipValue;
      }  
      for(int j = 0; j < NUM_DIMENSIONS; j++){
          newClusters[i*NUM_DIMENSIONS + j] = numerator[j]/denominator;

      }  
    }
    

    /*
    memset(newClusters,0.0,sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);    




    memset(denominators,0.0,sizeof(float)*NUM_CLUSTERS);    

    for(int i = 0; i < NUM_EVENTS; i++){
        for(int j = 0; j < NUM_DIMENSIONS; j++)
            numerator[j] = 0;

        // Compute distance from this event to each cluster
        for(int j = 0; j < NUM_CLUSTERS; j++){
            distances[j] = CalculateDistanceCPU(oldClusters,events,j,i);
        }

        // Find sum of all distances
        sum = 0.0;
        for(int j = 0; j < NUM_CLUSTERS; j++) {
            sum += distances[j];
        }

        for(int j = 0; j < NUM_CLUSTERS; j++){
            membershipValue = distances[j] / sum;
            //printf("%f\n",membershipValue);
            if(isnan(membershipValue)) {
                printf("Event #%d: MembershipValue: %f, sum: %f\n",i,membershipValue,sum);
            }

            // Add contribution to the center for each dimension for this cluster
            for(int k = 0; k < NUM_DIMENSIONS; k++){
              newClusters[j*NUM_DIMENSIONS+k] += events[i*NUM_DIMENSIONS + k]*membershipValue;
            }

            denominators[j] += membershipValue;
        }  
    }
    for(int k = 0; k < NUM_CLUSTERS; k++){
        for(int j = 0; j < NUM_DIMENSIONS; j++) {
            newClusters[k*NUM_DIMENSIONS + j] /= denominators[k];
            //printf("%f ",newClusters[k*NUM_DIMENSIONS + j]);
        }
        //printf("\n");
    } 
    //printf("\n"); 
    */
    
    free(numerator);
    free(denominators);
    free(distances);
}




float* ParseSampleInput(const char* filename){
    FILE* myfile = fopen(filename, "r");
    if(myfile == NULL){
        printf("Error: File DNE\n");
        return NULL;
    }
    char myline[1024];
    
    float* retVal = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    myfile = fopen(filename, "r");
#if !LINE_LABELS

    for(int i = 0; i < NUM_EVENTS; i++){
        fgets(myline, 1024, myfile);
        retVal[i*NUM_DIMENSIONS] = (float)atof(strtok(myline, DELIMITER));
        for(int j = 1; j < NUM_DIMENSIONS; j++){
            retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
        }
    }
#else
    fgets(myline, 1024, myfile);
    for(int i = 0; i < NUM_EVENTS; i++){
        fgets(myline, 1024, myfile);
        strtok(myline, DELIMITER);
        for(int j = 0; j < NUM_DIMENSIONS; j++){
            retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
        }
    }
#endif
    
    fclose(myfile);
    
    
    return retVal;
}

void FreeMatrix(float* d_matrix){
    CUDA_SAFE_CALL(hipFree(d_matrix));
}

float* BuildQGPU(float* d_events, float* d_clusters, float* distanceMatrix, float* mdlTime, int gpu_id, int num_gpus){
    float* d_matrix;
    int size = sizeof(float) * NUM_CLUSTERS*NUM_CLUSTERS;

    cudaTimer_t timer_gpu;
    cudaTimer_t timer_memcpy;
    createTimer(&timer_gpu);
    createTimer(&timer_memcpy);
    
    startTimer(timer_memcpy);
    hipMalloc((void**)&d_matrix, size);
    printCudaError();
    stopTimer(timer_memcpy);
    
    startTimer(timer_gpu);
    dim3 grid(NUM_CLUSTERS / num_gpus, NUM_CLUSTERS);
    int start_row = gpu_id*(NUM_CLUSTERS/num_gpus);
    printf("GPU %d: Starting row for Q Matrix: %d\n",gpu_id,start_row);

    printf("Launching Q Matrix Kernel\n");
    CalculateQMatrixGPUUpgrade<<<grid, Q_THREADS>>>(d_events, d_clusters, d_matrix, distanceMatrix, start_row);
    hipDeviceSynchronize();
    printCudaError();
    stopTimer(timer_gpu);

    startTimer(timer_memcpy);
    float* matrix = (float*)malloc(size);
    printf("Copying results to CPU\n");
    hipError_t error = hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");
    stopTimer(timer_memcpy);

    stopTimer(timer_gpu);
    *mdlTime = getTimerValue(timer_gpu);
    printf("Processing time for MDL GPU: %f (ms) \n", *mdlTime);
    printf("Memcpy time for MDL GPU: %f (ms) \n", getTimerValue(timer_memcpy));
    
    deleteTimer(timer_gpu);
    deleteTimer(timer_memcpy);
        
    printCudaError();
    
    FreeMatrix(d_matrix);
    return matrix;
}

/*float FindScoreGPU(float* d_matrix, long config){
    float* d_score;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_score, sizeof(float)));
    EvaluateSolutionGPU<<<1, 1>>>(d_matrix, config, d_score);
    float* score = (float*)malloc(sizeof(float));
    CUDA_SAFE_CALL(hipMemcpy(score, d_score, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(d_score));
    return *score;
}*/

