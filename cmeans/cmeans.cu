#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>

#include "cmeans.h"
#include "cmeans_kernel.cu"
#include "MDL.h"
#include "timers.h"

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else

void printCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }
}

bool InitCUDA(void)
{
    int count = 0;
    int i = 0;
    int device = -1;
    int num_procs = 0;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    printf("There are %d devices.\n",count);
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            printf("Device #%d - %s, Version: %d.%d\n",i,prop.name,prop.major,prop.minor);
            // Check if CUDA capable device
            if(prop.major >= 1) {
                if(prop.multiProcessorCount > num_procs) {
                    device = i;
                    num_procs = prop.multiProcessorCount;
                }
            }
        }
    }
    if(device == -1) {
        fprintf(stderr, "There is no device supporting CUDA.\n");
        return false;
    }

    device = DEVICE;
    printf("Using Device %d\n",device);
    CUDA_SAFE_CALL(hipSetDevice(device));

    DEBUG("CUDA initialized.\n");
    return true;
}

#endif

unsigned int timer_io; // Timer for I/O, such as reading FCS file and outputting result files
unsigned int timer_memcpy; // Timer for GPU <---> CPU memory copying
unsigned int timer_cpu; // Timer for processing on CPU
unsigned int timer_gpu; // Timer for kernels on the GPU
unsigned int timer_total; // Total time

/************************************************************************/
/* C-means Main                                                            */
/************************************************************************/
int main(int argc, char* argv[])
{
   
    CUT_SAFE_CALL(cutCreateTimer(&timer_io));
    CUT_SAFE_CALL(cutCreateTimer(&timer_memcpy));
    CUT_SAFE_CALL(cutCreateTimer(&timer_cpu));
    CUT_SAFE_CALL(cutCreateTimer(&timer_gpu));
    CUT_SAFE_CALL(cutCreateTimer(&timer_total));
    
    CUT_SAFE_CALL(cutStartTimer(timer_total));
    CUT_SAFE_CALL(cutStartTimer(timer_io));
    
    // [program name] [data file]
    if(argc != 2){
        printf("Usage: %s data.csv\n",argv[0]);
        return 1;
    }

    DEBUG("Parsing input file\n");
    float* myEvents = ParseSampleInput(argv[1]);
    
    if(myEvents == NULL){
        printf("Error reading input file. Exiting.\n");
        return 1;
    }
     
    DEBUG("Finished parsing input file\n");
    
    if(!InitCUDA()) {
        return 0;
    }
   
    // Seed random generator, used for choosing initial cluster centers 
    srand((unsigned)(time(0)));
    //srand(42);
    
    float* myClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    float* newClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    
    CUT_SAFE_CALL(cutStopTimer(timer_io));
    CUT_SAFE_CALL(cutStartTimer(timer_cpu));
    
    clock_t total_start;
    total_start = clock();

    // Select random cluster centers
    DEBUG("Randomly choosing initial cluster centers.\n");
    generateInitialClusters(myClusters, myEvents);
    
    // Transpose the events matrix
    // Threads within a block access consecutive events, not consecutive dimensions
    // So we need the data aligned this way for coaelsced global reads for event data
    DEBUG("Transposing data matrix.\n");
    float* transposedEvents = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    for(int i=0; i<NUM_EVENTS; i++) {
        for(int j=0; j<NUM_DIMENSIONS; j++) {
            transposedEvents[j*NUM_EVENTS+i] = myEvents[i*NUM_DIMENSIONS+j];
        }
    }
    
    float* memberships = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_EVENTS); 
    CUT_SAFE_CALL(cutStopTimer(timer_cpu));
    
    CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
    DEBUG("Allocating memory on GPU.\n");
    float* d_distanceMatrix;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_distanceMatrix, sizeof(float)*NUM_EVENTS*NUM_CLUSTERS));
    float* d_memberships;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_memberships, sizeof(float)*NUM_EVENTS*NUM_CLUSTERS));
    float* d_E;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_E, sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS));
    float* d_C;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_C, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
    float* d_nC;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
    
    DEBUG("Copying input data to GPU.\n");
    int size = sizeof(float)*NUM_DIMENSIONS*NUM_EVENTS;
    //CUDA_SAFE_CALL(hipMemcpy(d_E, myEvents, size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_E, transposedEvents, size, hipMemcpyHostToDevice));
    
    DEBUG("Copying initial cluster centers to GPU.\n");
    size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;
    CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
    CUT_SAFE_CALL(cutStopTimer(timer_memcpy));
    
    float diff;
    clock_t cpu_start, cpu_stop;
    cpu_start = clock();
    PRINT("Starting C-means\n");
    float averageTime = 0;
    int iterations = 0;
    do{
#if CPU_ONLY
        CUT_SAFE_CALL(cutStartTimer(timer_cpu));
        clock_t cpu_start, cpu_stop;
        cpu_start = clock();

        DEBUG("Starting UpdateCenters kernel.\n");
        UpdateClusterCentersCPU(myClusters, myEvents, newClusters);

        cpu_stop = clock();
        DEBUG("Processing tiem for CPU: %f (ms) \n", (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC)*(float)1e3);
        averageTime += (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC)*(float)1e3;
        CUT_SAFE_CALL(cutStopTimer(timer_cpu));
#else
        
        unsigned int timer = 0;
        CUT_SAFE_CALL(cutCreateTimer(&timer));
        CUT_SAFE_CALL(cutStartTimer(timer));

        size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;

        CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
        CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
        CUT_SAFE_CALL(cutStopTimer(timer_memcpy));
        
        int num_blocks_distance = NUM_EVENTS / NUM_THREADS_DISTANCE;
        if(NUM_EVENTS % NUM_THREADS_DISTANCE) {
            num_blocks_distance++;
        }
        int num_blocks_membership = NUM_EVENTS / NUM_THREADS_MEMBERSHIP;
        if(NUM_EVENTS % NUM_THREADS_DISTANCE) {
            num_blocks_membership++;
        }

        CUT_SAFE_CALL(cutStartTimer(timer_gpu));
        DEBUG("Launching ComputeDistanceMatrix kernel\n");
        //ComputeDistanceMatrix<<< NUM_CLUSTERS, NUM_THREADS_DISTANCE >>>(d_C, d_E, d_distanceMatrix);
        //ComputeDistanceMatrix2<<< dim3(NUM_CLUSTERS,num_blocks_distance), NUM_THREADS_DISTANCE >>>(d_C, d_E, d_distanceMatrix);
        ComputeDistanceMatrix3<<< dim3(num_blocks_distance,NUM_CLUSTERS), NUM_THREADS_DISTANCE >>>(d_C, d_E, d_distanceMatrix);
        hipDeviceSynchronize();
        printCudaError();
        
        DEBUG("Launching ComputeMembershipMatrix kernel\n");
        //ComputeMembershipMatrix<<< NUM_CLUSTERS, NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix, d_memberships);
        //ComputeMembershipMatrix2<<< dim3(NUM_CLUSTERS,num_blocks), NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix, d_memberships);
        ComputeMembershipMatrix3<<< dim3(num_blocks_membership,NUM_CLUSTERS), NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix, d_memberships);
        //ComputeMembershipMatrix4<<< dim3(NUM_CLUSTERS,num_blocks), NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix, d_memberships);
        hipDeviceSynchronize();
        printCudaError();
        DEBUG("Launching UpdateClusterCentersGPU kernel\n");
        //UpdateClusterCentersGPU<<< NUM_CLUSTERS, NUM_THREADS >>>(d_C, d_E, d_nC, d_distanceMatrix);
        UpdateClusterCentersGPU2<<< dim3(NUM_CLUSTERS,NUM_DIMENSIONS), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_memberships);
                
        hipDeviceSynchronize();
        DEBUG(hipGetErrorString(hipGetLastError()));
        DEBUG("\n");
        CUT_SAFE_CALL(cutStopTimer(timer_gpu));

        CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
        DEBUG("Copying centers from GPU\n");
        CUDA_SAFE_CALL(hipMemcpy(newClusters, d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS, hipMemcpyDeviceToHost));
        CUT_SAFE_CALL(cutStopTimer(timer_memcpy));
        
        CUT_SAFE_CALL(cutStopTimer(timer));
        float thisTime = cutGetTimerValue(timer);
        DEBUG("Processing time for GPU: %f (ms) \n", thisTime);
        averageTime += thisTime;
        CUT_SAFE_CALL(cutDeleteTimer(timer));

#endif

        CUT_SAFE_CALL(cutStartTimer(timer_cpu));
        
        diff = 0.0;
        for(int i=0; i < NUM_CLUSTERS; i++){
            DEBUG("Center %d: ",i);     
            for(int k = 0; k < NUM_DIMENSIONS; k++){
                DEBUG("%.2f ",newClusters[i*NUM_DIMENSIONS + k]);
                diff += fabs(myClusters[i*NUM_DIMENSIONS + k] - newClusters[i*NUM_DIMENSIONS + k]);
                myClusters[i*NUM_DIMENSIONS + k] = newClusters[i*NUM_DIMENSIONS + k];
            }
            DEBUG("\n");
        }
        DEBUG("Iteration %d Diff = %f\n", iterations, diff);

        iterations++;
        
        CUT_SAFE_CALL(cutStopTimer(timer_cpu));

    } while((iterations < MIN_ITERS) || (abs(diff) > THRESHOLD && iterations < MAX_ITERS)); 
  
    //CUT_SAFE_CALL(cutStartTimer(timer_gpu));
    ComputeNormalizedMembershipMatrix<<< NUM_CLUSTERS, NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix, d_memberships); 
    //CUT_SAFE_CALL(cutStopTimer(timer_gpu));
    CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
    DEBUG("Copying memberships from GPU\n");
    CUDA_SAFE_CALL(hipMemcpy(memberships,d_memberships,sizeof(float)*NUM_CLUSTERS*NUM_EVENTS,hipMemcpyDeviceToHost)); 
    CUT_SAFE_CALL(cutStopTimer(timer_memcpy));

    if(iterations == MAX_ITERS){
        PRINT("Warning: Did not converge to the %f threshold provided\n", THRESHOLD);
    }
    cpu_stop = clock();
    
    CUT_SAFE_CALL(cutStartTimer(timer_io));
    
    averageTime /= iterations;
    printf("\nTotal Processing time: %f (s) \n", (float)(cpu_stop - cpu_start)/(float)(CLOCKS_PER_SEC));
    printf("\n");

    CUT_SAFE_CALL(cutStopTimer(timer_io));
    
    int* finalClusterConfig;
    float mdlTime = 0;

    #if ENABLE_MDL 
        #if CPU_ONLY
            finalClusterConfig = MDL(myEvents, myClusters, &mdlTime, argv[1]);
        #else
            finalClusterConfig = MDLGPU(d_E, d_nC, d_distanceMatrix, &mdlTime, argv[1]);
            mdlTime /= 1000.0; // CUDA timer returns time in milliseconds, normalize to seconds
        #endif
    #else
        finalClusterConfig = (int*) malloc(sizeof(int)*NUM_CLUSTERS);
        memset(finalClusterConfig,1,sizeof(int)*NUM_CLUSTERS);
    #endif

    CUT_SAFE_CALL(cutStartTimer(timer_io));

    // Filters out the final clusters (Based on MDL)
    PRINT("Final Clusters are:\n");
    int newCount = 0;
    for(int i = 0; i < NUM_CLUSTERS; i++){
        if(finalClusterConfig[i]){
            for(int j = 0; j < NUM_DIMENSIONS; j++){
                newClusters[newCount * NUM_DIMENSIONS + j] = myClusters[i*NUM_DIMENSIONS + j];
                PRINT("%.2f\t", myClusters[i*NUM_DIMENSIONS + j]);
            }
            newCount++;
            PRINT("\n");
        }
    }
  
    #if ENABLE_OUTPUT 
        ReportSummary(newClusters, newCount, argv[1]);
        ReportResults(myEvents, memberships, newCount, argv[1]);
    #endif
    
    CUT_SAFE_CALL(cutStopTimer(timer_io));
    
    free(newClusters);
    free(myClusters);
    free(myEvents);
#if !CPU_ONLY
    CUDA_SAFE_CALL(hipFree(d_E));
    CUDA_SAFE_CALL(hipFree(d_C));
    CUDA_SAFE_CALL(hipFree(d_nC));
#endif

    CUT_SAFE_CALL(cutStopTimer(timer_total));
    printf("\n\n"); 
    printf("Total Time (ms): %f\n",cutGetTimerValue(timer_total));
    printf("I/O Time (ms): %f\n",cutGetTimerValue(timer_io));
    printf("CPU processing Time (ms): %f\n",cutGetTimerValue(timer_cpu));
    printf("GPU processing Time (ms): %f\n",cutGetTimerValue(timer_gpu));
    printf("GPU memcpy Time (ms): %f\n",cutGetTimerValue(timer_memcpy));
    printf("\n\n"); 
    return 0;
}

void generateInitialClusters(float* clusters, float* events){
    int seed;
    for(int i = 0; i < NUM_CLUSTERS; i++){
        seed = rand() % NUM_EVENTS;
        for(int j = 0; j < NUM_DIMENSIONS; j++){
            clusters[i*NUM_DIMENSIONS + j] = events[seed*NUM_DIMENSIONS + j];
        }
    }
    
}

__host__ float CalculateDistanceCPU(const float* clusters, const float* events, int clusterIndex, int eventIndex){

    float sum = 0;
#if DISTANCE_MEASURE == 0
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i];
        sum += tmp*tmp;
    }
    sum = sqrt(sum);
#endif
#if DISTANCE_MEASURE == 1
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i];
        sum += abs(tmp);
    }
#endif
#if DISTANCE_MEASURE == 2
    for(int i = 0; i < NUM_DIMENSIONS; i++){
        float tmp = abs(events[eventIndex*NUM_DIMENSIONS + i] - clusters[clusterIndex*NUM_DIMENSIONS + i]);
        if(tmp > sum)
            sum = tmp;
    }
#endif
    return sum;
}


__host__ float MembershipValue(const float* clusters, const float* events, int clusterIndex, int eventIndex){
    float myClustDist = CalculateDistanceCPU(clusters, events, clusterIndex, eventIndex);
    float sum =0;
    float otherClustDist;
    for(int j = 0; j< NUM_CLUSTERS; j++){
        otherClustDist = CalculateDistanceCPU(clusters, events, j, eventIndex); 
        if(otherClustDist < .000001)
            return 0.0;
        sum += pow((float)(myClustDist/otherClustDist),float(2/(FUZZINESS-1)));
    }
    return 1/sum;
}



void UpdateClusterCentersCPU(const float* oldClusters, const float* events, float* newClusters){
    
    
    //float membershipValue, sum, denominator;
    float membershipValue, denominator;
    float* numerator = (float*)malloc(sizeof(float)*NUM_DIMENSIONS);
    float* denominators = (float*)malloc(sizeof(float)*NUM_CLUSTERS);
    float* distances = (float*)malloc(sizeof(float)*NUM_CLUSTERS);

    
    for(int i = 0; i < NUM_CLUSTERS; i++){
      denominator = 0.0;
      for(int j = 0; j < NUM_DIMENSIONS; j++)
        numerator[j] = 0;
      for(int j = 0; j < NUM_EVENTS; j++){
        membershipValue = MembershipValue(oldClusters, events, i, j);
        for(int k = 0; k < NUM_DIMENSIONS; k++){
          numerator[k] += events[j*NUM_DIMENSIONS + k]*membershipValue;
        }
        
        denominator += membershipValue;
      }  
      for(int j = 0; j < NUM_DIMENSIONS; j++){
          newClusters[i*NUM_DIMENSIONS + j] = numerator[j]/denominator;
      }  
    }
    
    free(numerator);
    free(denominators);
    free(distances);
}


float* ParseSampleInput(const char* filename){
    FILE* myfile = fopen(filename, "r");
    if(myfile == NULL){
        printf("Error: File DNE\n");
        return NULL;
    }
    char myline[1024];
    
    float* retVal = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    myfile = fopen(filename, "r");
    #if LINE_LABELS
        fgets(myline, 1024, myfile);
        for(int i = 0; i < NUM_EVENTS; i++){
            fgets(myline, 1024, myfile);
            retVal[i*NUM_DIMENSIONS] = (float)atof(strtok(myline, DELIMITER));
            for(int j = 1; j < NUM_DIMENSIONS; j++){
                retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
            }
        }
    #else
        for(int i = 0; i < NUM_EVENTS; i++){
            fgets(myline, 1024, myfile);
            retVal[i*NUM_DIMENSIONS] = (float)atof(strtok(myline, DELIMITER));
            for(int j = 1; j < NUM_DIMENSIONS; j++){
                retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
            }
        }
    #endif
    
    fclose(myfile);
    
    
    return retVal;
}

void FreeMatrix(float* d_matrix){
    CUDA_SAFE_CALL(hipFree(d_matrix));
}

float* BuildQGPU(float* d_events, float* d_clusters, float* d_distanceMatrix, float* mdlTime){
    float* d_matrix;
    int size = sizeof(float) * NUM_CLUSTERS*NUM_CLUSTERS;

    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));
    
    CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
    hipMalloc((void**)&d_matrix, size);
    printCudaError();
    CUT_SAFE_CALL(cutStopTimer(timer_memcpy));
    CUT_SAFE_CALL(cutStartTimer(timer_gpu));

    dim3 grid(NUM_CLUSTERS, NUM_CLUSTERS);
    printf("Launching Q Matrix Kernel\n");
    CalculateQMatrixGPUUpgrade<<<grid, Q_THREADS>>>(d_events, d_clusters, d_matrix, d_distanceMatrix);
    hipDeviceSynchronize();
    printCudaError();

    CUT_SAFE_CALL(cutStopTimer(timer_gpu));
    

    CUT_SAFE_CALL(cutStartTimer(timer_memcpy));
    float* matrix = (float*)malloc(size);
    printf("Copying results to CPU\n");
    hipError_t error = hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printCudaError();
    CUT_SAFE_CALL(cutStopTimer(timer_memcpy));

    CUT_SAFE_CALL(cutStopTimer(timer));
    *mdlTime = cutGetTimerValue(timer);
    printf("Processing time for GPU: %f (ms) \n", *mdlTime);
    CUT_SAFE_CALL(cutDeleteTimer(timer));
        
    FreeMatrix(d_matrix);

    printf("Q Matrix:\n");
    for(int row=0; row < NUM_CLUSTERS; row++) {
        for(int col=0; col < NUM_CLUSTERS; col++) {
            printf("%f ",matrix[row*NUM_CLUSTERS+col]);
        }
        printf("\n");
    }
    return matrix;
}

