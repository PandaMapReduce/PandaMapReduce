#include "hip/hip_runtime.h"
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	Code Name: Panda 0.1
	File: reduce.cu 
	Time: 2012-07-01 
	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.
 
 */

#ifndef __REDUCE_CU__
#define __REDUCE_CU__

#include "Panda.h"
#include "UserAPI.h"

//invoke cmeans_cpu_map_cpp compiled with g++
void cpu_map(void *key, void *val, int keySize, int valSize, cpu_context *d_g_state, int map_task_idx){
	
	cmeans_cpu_map_cpp(key, val, keySize, valSize);
	CPUEmitMapOutput(key, val, sizeof(KM_KEY_T), sizeof(KM_VAL_T), d_g_state, map_task_idx);

}


void cpu_map2(void *key, void *val, int keySize, int valSize, cpu_context *d_g_state, int map_task_idx){

	KM_KEY_T* pKey = (KM_KEY_T*)key;
	KM_VAL_T* pVal = (KM_VAL_T*)val;
	
	int dim = pKey->dim;
	//int dim_4;
	int K = pKey->K;
	int start = pKey->start;
	int end = pKey->end;
	int index = pKey->local_map_id;
	//TODO there could be problem here when running C-means with more than one GPU
	//index = 0;

	float *point	= (float*)(pVal->d_Points);
	float *cluster	= (float*)(pVal->d_Clusters);

	float * tempClusters = pVal->d_tempClusters+index*dim*K;
	float * tempDenominators = pVal->d_tempDenominators+index*K;

	float denominator = 0.0f;
	float membershipValue = 0.0f;

	float *distances = (float *)malloc(sizeof(float)*K);
	float *numerator = (float *)malloc(sizeof(float)*K);
	
	for(int i=0; i<K; i++){
		distances[i]=0.0f;
		numerator[i]=0.0f;
	}//for

	//printf("map_task_id 0:%d thread_id:%d\n",map_task_idx,THREAD_ID);
	for (int i=start; i<end; i++){
		float *curPoint = (float*)(pVal->d_Points + i*dim);
		for (int k = 0; k < K; ++k)
		{
			float* curCluster = (float*)(pVal->d_Clusters + k*dim);
			distances[k] = 0.0;
			//printf("dim:%d\n",dim);
			//dim_4 = dim;
			float delta = 0.0;	
			
			for (int j = 0; j < dim; ++j)
			{
				delta = curPoint[j]-curCluster[j];
				distances[k] += (delta*delta);
			}//for
			
			numerator[k] = powf(distances[k],2.0f/(2.0-1.0))+1e-30;
			denominator  = denominator + 1.0f/(numerator[k]+1e-30);
		}//for

		for (int k = 0; k < K; ++k)
		{
			membershipValue = 1.0f/powf(numerator[k]*denominator,(float)2.0);
			for(int d =0; d<dim; d++){
				//float pt = curePoint[d].x;
				tempClusters[k*dim+d] += (curPoint[d])*membershipValue;
				
			}
			tempDenominators[k]+= membershipValue;
		}//for 
	}//for
	//printf("map_task_id 1:%d\n",map_task_idx);
	
	free(distances);
	free(numerator);
	
	//TODO
	pKey->local_map_id = 0;
	pKey->end = 0;
	pKey->start = 0;
	pKey->global_map_id = 0;
	
	CPUEmitMapOutput(key, val, sizeof(KM_KEY_T), sizeof(KM_VAL_T), d_g_state, map_task_idx);

}//void

__device__ void gpu_map(void *key, void *val, int keySize, int valSize, gpu_context *d_g_state, int map_task_idx){

	KM_KEY_T* pKey = (KM_KEY_T*)key;
	KM_VAL_T* pVal = (KM_VAL_T*)val;
	
	int dim = pKey->dim;
	int dim_4;
	int K = pKey->K;
	int start = pKey->start;
	int end = pKey->end;
	int index = pKey->local_map_id;
	//TODO there could be problem here when running C-means with more than one GPU
	index = 0;

	float4 *point =(float4*)(pVal->d_Points);
	float* cluster = (float*)(pVal->d_Clusters);

	float * tempClusters = pVal->d_tempClusters+index*dim*K;
	float * tempDenominators = pVal->d_tempDenominators+index*K;

	float denominator = 0.0f;
	float membershipValue = 0.0f;

	float *distances = (float *)malloc(sizeof(float)*K);
	float *numerator = (float *)malloc(sizeof(float)*K);
	
	
	for(int i=0; i<K; i++){
		distances[i]=0.0f;
		numerator[i]=0.0f;
	}//for

	//printf("map_task_id 0:%d thread_id:%d\n",map_task_idx,THREAD_ID);
	for (int i=start; i<end; i++){
		float4* curPoint = (float4*)(pVal->d_Points + i*dim);
		for (int k = 0; k < K; ++k)
		{
			float4* curCluster = (float4*)(pVal->d_Clusters + k*dim);
			distances[k] = 0.0;
			//printf("dim:%d\n",dim);
			dim_4 = dim/4;
			float delta = 0.0;	
			
			for (int j = 0; j < dim_4; ++j)
			{
				float4 pt = curPoint[j];
				float4 cl = curCluster[j];

				delta = pt.x-cl.x;
				distances[k] += (delta*delta);
				delta = pt.y-cl.y;
				distances[k] += (delta*delta);
				delta = pt.z-cl.z;
				distances[k] += (delta*delta);
				delta = pt.w-cl.w;
				distances[k] += (delta*delta);

			}//for
				
			int remainder = dim & 0x00000003;
			float* rPoint = (float*)(curPoint+dim_4);
			float* rCluster = (float*)(curCluster+dim_4);
			
			for (int j = 0; j < remainder; j++)
			{
				float pt = rPoint[j];
				float cl = rCluster[j];
				delta = pt - cl;
				distances[k] += (delta*delta);				
			}			
			numerator[k] = powf(distances[k],2.0f/(2.0-1.0))+1e-30;
			denominator  = denominator + 1.0f/(numerator[k]+1e-30);
		}//for

		for (int k = 0; k < K; ++k)
		{
			membershipValue = 1.0f/powf(numerator[k]*denominator,(float)2.0);
			for(int d =0;d<dim_4;d++){
				//float pt = curePoint[d].x;
				tempClusters[k*dim+d] += (curPoint[d].x)*membershipValue;
				tempClusters[k*dim+d] += (curPoint[d].y)*membershipValue;
				tempClusters[k*dim+d] += (curPoint[d].z)*membershipValue;
				tempClusters[k*dim+d] += (curPoint[d].w)*membershipValue;
			}
			tempDenominators[k]+= membershipValue;
		}//for 
	}//for
	//printf("map_task_id 1:%d\n",map_task_idx);
	
	free(distances);
	free(numerator);
	
	//TODO
	pKey->local_map_id = 0;
	pKey->end = 0;
	pKey->start = 0;
	pKey->global_map_id = 0;
	
	GPUEmitMapOutput(key, val, sizeof(KM_KEY_T), sizeof(KM_VAL_T), d_g_state, map_task_idx);
	
}//map2





__device__ int gpu_compare(const void *key_a, int len_a, const void *key_b, int len_b)
{
	//KM_KEY_T *ka = (KM_KEY_T*)key_a;
	//KM_KEY_T *kb = (KM_KEY_T*)key_b;

	return 0;

	/*
	if (ka->i > kb->i)
		return 1;

	if (ka->i > kb->i)
		return -1;

	if (ka->i == kb->i)
		return 0;
	*/
}


int cpu_compare(const void *key_a, int len_a, const void *key_b, int len_b)
{
	//KM_KEY_T *ka = (KM_KEY_T*)key_a;
	//KM_KEY_T *kb = (KM_KEY_T*)key_b;

	return 0;

	/*
	if (ka->i > kb->i)
		return 1;

	if (ka->i > kb->i)
		return -1;

	if (ka->i == kb->i)
		return 0;
		*/

}


void cpu_reduce(void *key, val_t* vals, int keySize, int valCount, cpu_context* d_g_state){
	cmeans_cpu_reduce_cpp(key,  vals, keySize, valCount);
	CPUEmitReduceOutput(key,vals,sizeof(KM_KEY_T), sizeof(KM_VAL_T), d_g_state);
}

//-------------------------------------------------------------------------
//Reduce Function in this application
//-------------------------------------------------------------------------

void cpu_reduce2(void *key, val_t* vals, int keySize, int valCount, cpu_context* d_g_state)
{

		KM_KEY_T* pKey = (KM_KEY_T*)key;
        int dim = pKey->dim;
        int K = pKey->K;



        float* myClusters = (float*) malloc(sizeof(float)*dim*K);
        float* myDenominators = (float*) malloc(sizeof(float)*K);
        memset(myClusters,0,sizeof(float)*dim*K);
        memset(myDenominators,0,sizeof(float)*K);

        float *tempClusters = NULL;
        float *tempDenominators = NULL;
		

        for (int i = 0; i < valCount; i++)
        {
                int index = pKey->local_map_id;


				KM_VAL_T* pVal = (KM_VAL_T*)(vals[i].val);
                tempClusters = pVal->d_tempClusters + index*K*dim;
                tempDenominators = pVal->d_tempDenominators+ index*K;
                for (int k = 0; k< K; k++){
                        for (int j = 0; j< dim; j++)
                                myClusters[k*dim+j] += tempClusters[k*dim+j];
                        myDenominators[k] += tempDenominators[k];
                }//for
        }//end for


        for (int k = 0; k< K; k++){
			for (int i = 0; i < dim; i++){
						//printf("K:%d dim:%d myDenominators[i]:%f",K,dim,myDenominators[i]);
                        myClusters[i] /= ((float)myDenominators[i]+0.0001);
						//printf("%f ",myClusters[i]);
			}//for
			//printf("\n");
        }//for

		
		free(myClusters);
		free(myDenominators);

		CPUEmitReduceOutput(key,vals,sizeof(KM_KEY_T), sizeof(KM_VAL_T), d_g_state);

}

__device__ void gpu_combiner(void *KEY, val_t* VAL, int keySize, int valCount, gpu_context *d_g_state, int map_task_idx){
		
		
}//reduce2

void cpu_combiner(void *KEY, val_t* VAL, int keySize, int valCount, cpu_context *d_g_state, int map_task_idx){
		
		
}//reduce2

__device__ void gpu_reduce(void *key, val_t* vals, int keySize, int valCount, gpu_context d_g_state)
{
		//printf("valCount:%d\n",valCount);
		KM_KEY_T* pKey = (KM_KEY_T*)key;
        //KM_VAL_T* pVal = (KM_VAL_T*)vals;
        int dim = pKey->dim;
        int K = pKey->K;
				
        float* myClusters = (float*) malloc(sizeof(float)*dim*K);
        float* myDenominators = (float*) malloc(sizeof(float)*K);
        memset(myClusters,0,sizeof(float)*dim*K);
        memset(myDenominators,0,sizeof(float)*K);

        float *tempClusters = NULL;
        float *tempDenominators = NULL;
        for (int i = 0; i < valCount; i++)
        {
                int index = pKey->local_map_id;
				KM_VAL_T* pVal = (KM_VAL_T*)(vals[i].val);
                tempClusters = pVal->d_tempClusters + index*K*dim;
                tempDenominators = pVal->d_tempDenominators+ index*K;
                for (int k = 0; k< K; k++){
                        for (int j = 0; j< dim; j++)
                                myClusters[k*dim+j] += tempClusters[k*dim+j];
                        myDenominators[k] += tempDenominators[k];
                }//for
        }//end for

        for (int k = 0; k< K; k++){
			for (int i = 0; i < dim; i++){
                        myClusters[i] /= (float)myDenominators[i];
						//printf("%f ",myClusters[i]);
			}//for
			//printf("\n");
        }//for

		//printf("TID reduce2:%d\n",TID);
		GPUEmitReduceOuput(key,vals,sizeof(KM_KEY_T), sizeof(KM_VAL_T), &d_g_state);
		
		free(myClusters);
		free(myDenominators);
				
}//reduce2

#endif //__REDUCE_CU__
