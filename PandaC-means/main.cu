/*	

Copyright 2012 The Trustees of Indiana University.  All rights reserved.
CGL MapReduce Framework on GPUs and CPUs
Code Name: Panda 0.2
File: main.cu 
Time: 2012-07-01 
Developer: Hui Li (lihui@indiana.edu)

This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

*/

#include "Panda.h"
#include "UserAPI.h"
#include <ctype.h>


//-----------------------------------------------------------------------
//usage: C-means datafile
//param: datafile 
//-----------------------------------------------------------------------


static float *GenPointsFloat(int numPt, int dim)
{
	float *matrix = (float*)malloc(sizeof(float)*numPt*dim);
	srand(time(0));
	for (int i = 0; i < numPt; i++)
		for (int j = 0; j < dim; j++)
			matrix[i*dim+j] = (float)((rand() % 100)/73.0);
	return matrix;
}//static float 

static float *GenInitCentersFloat(float* points, int numPt, int dim, int K)
{
	float* centers = (float*)malloc(sizeof(float)*K*dim);

	for (int i = 0; i < K; i++)
		for (int j = 0; j < dim; j++)
			centers[i*dim+j] = points[i*dim + j];
	return centers;
}//

int main(int argc, char** argv) 
{		
	if (argc != 7)
	{
		printf("Panda C-means\n");
		printf("usage: %s [numPt] [Dimensions] [numClusters] [numMapperPerGPU] [maxIter] [numGpus]\n", argv[0]);
		exit(-1);
	}//if

	//printf("start %s  %s  %s\n",argv[0],argv[1],argv[2]);
	int numPt = atoi(argv[1]);
	int dim = atoi(argv[2]);
	int K = atoi(argv[3]);
	int numMapper = atoi(argv[4]);
	int maxIter = atoi(argv[5]);
	int num_gpus = atoi(argv[6]);
	double ratio = 0;
	
	panda_context *panda = CreatePandaContext();
	panda->num_gpus = num_gpus;
	panda->num_cpus_groups = 0;//num_cpus_groups;
	panda->ratio = 0.0;

	ShowLog("numPt:%d	dim:%d	K:%d	numMapper:%d	maxIter:%d",numPt,dim,K,numMapper,maxIter);
	float* h_points = GenPointsFloat(numPt, dim);
	float* h_cluster = GenInitCentersFloat(h_points, numPt, dim, K);
	
	int numgpus = 0;
	hipGetDeviceCount(&numgpus);
	if (num_gpus >= numgpus)		num_gpus = numgpus;
			
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*num_gpus);
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*num_gpus);
		
	for (int i=0; i<num_gpus; i++){

		job_configuration *gpu_job_conf = CreateJobConf();
		gpu_job_conf->num_gpus = num_gpus;
		gpu_job_conf->num_mappers = numMapper;
		gpu_job_conf->auto_tuning = false;
		gpu_job_conf->ratio = (double)ratio;
		gpu_job_conf->auto_tuning_sample_rate = -1;//sample_rate;
		gpu_job_conf->iterative_support = false;
		
		int tid = i;		
		float* d_points	=	NULL;
		float* d_cluster =	NULL;
		int* d_change	=	NULL;
		int* d_clusterId =	NULL;
		
		float* d_tempClusters = NULL;
		float* d_tempDenominators = NULL;
		
		checkCudaErrors(hipSetDevice(tid));
				
		checkCudaErrors(hipMalloc((void**)&d_points, numPt*dim*sizeof(int)));
		checkCudaErrors(hipMemcpy(d_points, h_points, numPt*dim*sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&d_clusterId, numPt*sizeof(int)));
		checkCudaErrors(hipMemset(d_clusterId, 0, numPt*sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_cluster, K*dim*sizeof(int)));
		checkCudaErrors(hipMemcpy(d_cluster, h_cluster, K*dim*sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&d_change, sizeof(int)));
		checkCudaErrors(hipMemset(d_change, 0, sizeof(int)));
		
		checkCudaErrors(hipMalloc((void**)&d_tempClusters,K*dim*numMapper*sizeof(float)));
		checkCudaErrors(hipMemset(d_tempClusters, 0, sizeof(float)*K*dim*numMapper));
		checkCudaErrors(hipMalloc((void**)&d_tempDenominators,numMapper * K * sizeof(float)));
		
		checkCudaErrors(hipMemset(d_tempDenominators, 0, sizeof(float)*K*numMapper));
		
		thread_info[i].tid = i;
		//thread_info[i].num_gpus = num_gpus;
		thread_info[i].device_type = GPU_ACC;
		
		hipDeviceProp_t gpu_dev;
		hipGetDeviceProperties(&gpu_dev, i);

		ShowLog("Configure Device ID:%d: Device Name:%s", i, gpu_dev.name);
		thread_info[i].device_name = gpu_dev.name;
				
		KM_VAL_T val;
		val.ptrPoints = (int *)d_points;
		val.ptrClusters = (int *)d_cluster;
		val.d_Points = d_points;
		val.d_Clusters = d_cluster;
		val.ptrChange = d_change;
		
		KM_KEY_T key;
		key.dim = dim;
		key.K = K;
		key.ptrClusterId = d_clusterId;
		
		int numPtPerGPU = numPt/num_gpus;
		int start = i*numPtPerGPU;
		int end = start+numPtPerGPU;
		if (i==num_gpus-1)
			end = numPt;
		
		int numPtPerMap = (end-start)/numMapper;
		ShowLog("numPtPerMap:%d startPt:%d  endPt:%d numPt:%d",numPtPerMap,start,end,numPt);

		int start_i,end_i;
		start_i = start;
		for (int j = 0; j < numMapper; j++)
		{	
			end_i = start_i + numPtPerMap;
			if (i<(end-start)%numMapper)
				end_i++;
			
			//ShowLog("start_i:%d, start_j:%d",start_i,end_i);
			//key.point_id = start_i;
			key.start = start_i;
			key.end = end_i;
			key.global_map_id = i*numMapper+j;
			key.local_map_id = j;

			val.d_Points = d_points;
			val.d_tempDenominators = d_tempDenominators;
			val.d_tempClusters = d_tempClusters;

			AddPandaTask(gpu_job_conf, &key, &val, sizeof(KM_KEY_T), sizeof(KM_VAL_T));
			start_i = end_i;
		}//for

		thread_info[i].job_conf = gpu_job_conf;
		thread_info[i].device_type = GPU_ACC;
	}//for
	double t1 = PandaTimer();
		
	int iter = 0;
	while (iter<maxIter)
	{
		PandaMetaScheduler(thread_info, panda);
		/*for (int i=0; i<num_gpus; i++){
			if (pthread_create(&(no_threads[i]),NULL,Panda_Map,(char *)&(thread_info[i]))!=0) 
				perror("Thread creation failed!\n");
		}//for num_gpus
		for (int i=0; i<num_gpus; i++){
			void *exitstat;
			if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
		}//for
		int gpu_id;
		hipGetDevice(&gpu_id);
		ShowLog("current gpu_id:%d",gpu_id);
		if(gpu_id !=(num_gpus-1)){
			checkCudaErrors(hipSetDevice(num_gpus-1));
			ShowLog("changing GPU context to device:%d",num_gpus-1);
		}//if*/
		
		/*	for (int i=1; i<num_gpus; i++){
			Panda_Shuffle_Merge_GPU(thread_info[i-1].d_g_state, thread_info[i].d_g_state);
		}//for
		//hipDeviceSynchronize();
		Panda_Reduce(&thread_info[num_gpus-1]);	*/
		iter++;
		hipDeviceSynchronize();
	}//while iterations

	double t2 = PandaTimer();
	ShowLog("Panda C-means take %f sec", t2-t1);
	DoLog2Disk("== Panda C-means numPt:%d	dim:%d	K:%d	numMapper:%d	maxIter:%d take %f sec",numPt,dim,K,numMapper,maxIter, t2-t1);

	return 0;
}//		
