/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 
	
	File: main.cu 
	First Version:	2012-07-01 V0.1
	Current Version: V0.3	
	Last Updates:   2012-8-29

	Developer: Hui Li (lihui@indiana.edu)
	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */


#include "Panda.h"
#include "Global.h"
#include <ctype.h>

//-----------------------------------------------------------------------
//	Panda Matrix Multiplication
//	Data: 8/29/2012
//-----------------------------------------------------------------------

static float *GenMatrix(int M_ROW_COUNT, int M_COL_COUNT, float init)
{
        float *matrix = (float*)malloc(sizeof(float)*M_ROW_COUNT*M_COL_COUNT);
        //srand(time(0));
		//memset(matrix,1.0,sizeof(float)*M_ROW_COUNT*M_COL_COUNT);
		for (int i = 0; i < M_ROW_COUNT; i++)
                for (int j = 0; j < M_COL_COUNT; j++)
                        //matrix[i*M_COL_COUNT+j] = (float)(rand() % 100);
						matrix[i*M_COL_COUNT+j] = init;
		return matrix;
}

static float *RotateMatrix(float *matrix, int rowCount, int colCount)
{
        float *m = (float*)malloc(sizeof(float)*rowCount*colCount);
        for (int i = 0; i < rowCount; i++)
                for (int j = 0; j < colCount; j++)
                                m[i * colCount + j] = matrix[i + colCount * j];
        return m;
}//static float
		
int main(int argc, char** argv)
{		
	if (argc != 8)
	{	
		printf("usage: %s [rowNum][colNum][num gpus][num cpu groups][num_mappers][cpu/gpu work ratio][sample rate]\n", argv[0]);
		exit(-1);	
	}//if
	
	DoLog("configure input data for Panda job");

	int ROW_NUM = atoi(argv[1]);
	int COL_NUM = atoi(argv[2]);
	int num_gpus = atoi(argv[3]);
	int num_cpus_groups = atoi(argv[4]);
	int num_mappers = atoi(argv[5]);
	float ratio = atof(argv[6]);
	int sample_rate = atoi(argv[7]);

	double t1 = PandaTimer();

	float *matrix1 = GenMatrix(ROW_NUM,COL_NUM, 1.0);
	float *tmpMatrix2 = GenMatrix(COL_NUM,ROW_NUM, 1.0);
    float *matrix2 = RotateMatrix(tmpMatrix2,COL_NUM,ROW_NUM);
	float *matrix3 = GenMatrix(ROW_NUM,COL_NUM, 0.0);

	double t2 = PandaTimer();
	DoLog("load matrices  num_gpus:%d  num_cpus_groups:%d", num_gpus, num_cpus_groups);
	double t3 = PandaTimer();

	MM_KEY_T key;
    MM_VAL_T val;
	val.row_dim = ROW_NUM;
    val.col_dim = COL_NUM;
	val.bz = BLOCK_SIZE;

	int start_row_id, end_row_id;
	start_row_id = end_row_id = 0;

	int cpu_task_num = (int)(ROW_NUM*ratio);
	int gpu_task_num = ROW_NUM - cpu_task_num;
	//panda worker
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*(num_gpus + num_cpus_groups));

	for (int dev_id=0; dev_id<num_cpus_groups; dev_id++){
		if (cpu_task_num == 0)
			break;

		//panda job
		job_configuration *cpu_job_conf = GetJobConf();
		cpu_job_conf->num_cpus_groups = num_cpus_groups;
		cpu_job_conf->num_cpus_cores = getCPUCoresNum();

		int partitions = cpu_job_conf->num_cpus_cores*num_cpus_groups;
		int task_per_partition = ((cpu_task_num)/partitions);

		key.h_matrix1 = matrix1;
		key.h_matrix2 = matrix2;
		key.h_matrix3 = matrix3;

		key.matrix1 = NULL;
		key.matrix2 = NULL;
		key.matrix3 = NULL;

		for (int i=0;i<partitions;i++){
			start_row_id = task_per_partition*i;
			end_row_id = start_row_id+task_per_partition;
			val.row = start_row_id;
			val.col = end_row_id;
			if( i==(partitions-1) )
				val.col = cpu_task_num;

			if (val.col>val.row)
				AddPandaTask(cpu_job_conf, &key, &val, sizeof(MM_KEY_T), sizeof(MM_VAL_T));
		}//for

		thread_info[dev_id].job_conf = cpu_job_conf;
		thread_info[dev_id].device_type = CPU_ACC;

	}//for
		
	for (int dev_id=0; dev_id<num_gpus; dev_id++){

		job_configuration *gpu_job_conf = GetJobConf();
		gpu_job_conf->num_gpus = num_gpus;
		gpu_job_conf->num_mappers = num_mappers;
		gpu_job_conf->auto_tuning = false;
		gpu_job_conf->ratio = (double)ratio;
		gpu_job_conf->auto_tuning_sample_rate = sample_rate;

		if ( dev_id == 0 )
			start_row_id = cpu_task_num;	
		else 
			start_row_id = cpu_task_num+(dev_id)*(gpu_task_num/num_gpus);

		end_row_id = start_row_id + gpu_task_num/num_gpus;
		if ( dev_id == (num_gpus - 1) )
			end_row_id = cpu_task_num + gpu_task_num;

		//copy to data into different GPU device
		hipSetDevice(dev_id);  

		int matrixSize = sizeof(float)*ROW_NUM*COL_NUM;
		float *d_matrix1 = NULL;
		hipMalloc((void **)&d_matrix1,matrixSize);
		hipMemcpy(d_matrix1, matrix1, matrixSize, hipMemcpyHostToDevice);
		float *d_matrix2 = NULL;
		hipMalloc((void**)&d_matrix2,matrixSize);
		hipMemcpy(d_matrix2,matrix2,matrixSize,hipMemcpyHostToDevice);
		float *d_matrix3 = NULL;
		hipMalloc((void**)&d_matrix3,matrixSize);
		hipMemcpy(d_matrix3,matrix3,matrixSize,hipMemcpyHostToDevice);

		key.matrix1 = d_matrix1;
		key.matrix2 = d_matrix2;
		key.matrix3 = d_matrix3;

		key.h_matrix1 = NULL;
		key.h_matrix2 = NULL;
		key.h_matrix3 = NULL;

		if(end_row_id>start_row_id)
			for (int i = start_row_id/BLOCK_SIZE; i < (end_row_id + BLOCK_SIZE-1)/BLOCK_SIZE; i++)
			{
				val.row = i;
				for (int j = 0; j < (COL_NUM+BLOCK_SIZE-1)/BLOCK_SIZE; j++)
				{
					val.col = j;
					AddPandaTask(gpu_job_conf, &key, &val, sizeof(MM_KEY_T), sizeof(MM_VAL_T));
				}//for
			}//for
			thread_info[num_cpus_groups + dev_id].job_conf = gpu_job_conf;
			thread_info[num_cpus_groups + dev_id].device_type = GPU_ACC;

	}

	double t4 = PandaTimer();
	PandaMetaScheduler(thread_info, num_gpus, num_cpus_groups);
	hipDeviceSynchronize();
	double t5 = PandaTimer();

	DoLog("GenMatrix:%f",t2-t1);
	DoLog("Copy to GPU:%f",t4-t3);
	DoLog("Compute:%f",t5-t4);
	char str[128];
	sprintf(str,"matrix size:%d copy to GPU:%f  compute:%f cpu/gpu ratio:%f sample rate:%d", ROW_NUM, t3-t2, t5-t4, (double)ratio,sample_rate);
	DoDiskLog(str);

	return 0;
}//		