#include "hip/hip_runtime.h"
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	Code Name: Panda 0.1
	File: PandaUtils.cu 
	Time: 2012-07-01 
	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.
 
 */

#ifndef __PANDAUTILS_CU__
#define __PANDAUTILS_CU__

#include "Panda.h"




//--------------------------------------------------------
//start a timer
//
//param	: start_tv
//--------------------------------------------------------



void startTimer(TimeVal_t *start_tv)
{
   //gettimeofday((struct timeval*)start_tv, NULL);
}

//--------------------------------------------------------
//end a timer, and print out a message
//
//param	: msg message to print out
//param	: start_tv
//--------------------------------------------------------
void endTimer(char *msg, TimeVal_t *start_tv)
{
   /*hipDeviceSynchronize();
   struct timeval end_tv;

   gettimeofday(&end_tv, NULL);

   time_t sec = end_tv.tv_sec - start_tv->tv_sec;
   time_t ms = end_tv.tv_usec - start_tv->tv_usec;

   time_t diff = sec * 1000000 + ms;
	*/
   //printf("%10s:\t\t%fms\n", msg, (double)((double)diff/1000.0));
}//void


//----------------------------------------------------------
//print output records
//
//param: spec
//param: num -- maximum number of output records to print
//param: printFunc -- a function pointer
//	void printFunc(void* key, void* val, int keySize, int valSize)
//----------------------------------------------------------
void PrintOutputRecords(Spec_t* spec, int num, PrintFunc_t printFunc)
{
	/*
	int maxNum = num;
	if (maxNum > spec->outputRecordCount || maxNum < 0) maxNum = spec->outputRecordCount;
	for (int i = 0; i < maxNum; ++i)
	{
		int4 index = spec->outputOffsetSizes[i];
		printFunc((char*)spec->outputKeys + index.x, (char*)spec->outputVals + index.z, index.y, index.w);
	}
	*/
}//void

#endif //__PANDAUTILS_CU__