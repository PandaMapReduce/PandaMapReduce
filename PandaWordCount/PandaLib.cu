#include "hip/hip_runtime.h"
/*	
	Copyright 2012 The Trustees of Indiana University.  All rights reserved.
	CGL MapReduce Framework on GPUs and CPUs
	
	Code Name: Panda 0.20
	
	File: PandaLib.cu 
	First Version:	2012-07-01 V0.1
	Github: https://github.com/cyberaide/biostatistics/tree/master/GPUMapReduce			

	Developer: Hui Li (lihui@indiana.edu)

	This is the source code for Panda, a MapReduce runtime on GPUs and CPUs.

 */

#ifndef __PANDALIB_CU__
#define __PANDALIB_CU__

#include "Panda.h"
#include "stdlib.h"
#include "map.cu"
#include "reduce.cu"

//----------------------------------------------
//Get default runtime configuration
//return: default spec
//----------------------------------------------

job_configuration *GetJobConf(){

	job_configuration *job_conf = (job_configuration *)malloc(sizeof(job_configuration));

	if (job_conf == NULL) exit(-1);
	memset(job_conf, 0, sizeof(job_configuration));
	job_conf->num_input_record = 0;
	job_conf->input_keyval_arr = NULL;
	job_conf->auto_tuning = false;
	
	job_conf->num_mappers = 0;
	job_conf->num_reducers = 0;
	job_conf->num_gpus = 0;
	job_conf->num_cpus_cores = 0;
	job_conf->num_cpus_groups = 0;

	return job_conf;
}//gpu_context


gpu_context *GetGPUContext(){
	gpu_context *d_g_state = (gpu_context*)malloc(sizeof(gpu_context));
	if (d_g_state == NULL) exit(-1);
	memset(d_g_state, 0, sizeof(gpu_context));
	d_g_state->configured = false;
	d_g_state->h_input_keyval_arr = NULL;
	d_g_state->num_mappers = 0;
	d_g_state->num_reducers = 0;
	return d_g_state;
}//gpu_context
			 
cpu_context *GetCPUContext(){
	cpu_context *d_g_state = (cpu_context*)malloc(sizeof(cpu_context));
	if (d_g_state == NULL) exit(-1);
	memset(d_g_state, 0, sizeof(cpu_context));
	d_g_state->configured = false;
	d_g_state->input_keyval_arr = NULL;
	return d_g_state;
}//gpu_context

panda_context *GetPandaContext(){
	
	panda_context *d_g_state = (panda_context*)malloc(sizeof(panda_context));
	if (d_g_state == NULL) exit(-1);
	//memset(d_g_state, 0, sizeof(gpu_context));
	//d_g_state->configured = false;
	
	d_g_state->input_keyval_arr = NULL;
	d_g_state->intermediate_keyval_arr_arr_p = NULL;
	d_g_state->sorted_intermediate_keyvals_arr = NULL;
	d_g_state->sorted_keyvals_arr_len = 0;
	d_g_state->num_gpus = 0;
	d_g_state->gpu_context = NULL;
	d_g_state->num_cpus_groups = 0;
	d_g_state->cpu_context = NULL;
	return d_g_state;

}//gpu_context


//For version 0.3
void InitCPUMapReduce2(thread_info_t * thread_info){

	cpu_context *d_g_state = (cpu_context *)(thread_info->d_g_state);
	job_configuration *job_conf = (job_configuration *)(thread_info->job_conf);

	if (job_conf->num_input_record<=0) { DoLog("Error: no any input keys"); exit(-1);}
	if (job_conf->input_keyval_arr == NULL) { DoLog("Error: input_keyval_arr == NULL"); exit(-1);}
	if (d_g_state->num_cpus_cores <= 0) {	DoLog("Error: d_g_state->num_cpus == 0"); exit(-1);}

	//DoLog("d_g_state->configured:%s  enable for iterative applications",d_g_state->configured? "true" : "false");
	//if (d_g_state->configured)
	//	return;
	
	int totalKeySize = 0;
	int totalValSize = 0;

	for(int i=0;i<job_conf->num_input_record;i++){
		totalKeySize += job_conf->input_keyval_arr[i].keySize;
		totalValSize += job_conf->input_keyval_arr[i].valSize;
	}//for

	DoLog("d_g_state->num_input_record:%d, totalKeySize:%d totalValSize:%d num_cpus:%d", job_conf->num_input_record, totalKeySize, totalValSize, d_g_state->num_cpus_cores);

	//TODO determin num_cpus
	//d_g_state->num_cpus = 12;

	int num_cpus_cores = d_g_state->num_cpus_cores;

	d_g_state->panda_cpu_task = (pthread_t *)malloc(sizeof(pthread_t)*(num_cpus_cores));
	d_g_state->panda_cpu_task_info = (panda_cpu_task_info_t *)malloc(sizeof(panda_cpu_task_info_t)*(num_cpus_cores));

	d_g_state->intermediate_keyval_arr_arr_p = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*job_conf->num_input_record);
	memset(d_g_state->intermediate_keyval_arr_arr_p, 0, sizeof(keyval_arr_t)*job_conf->num_input_record);

	for (int i=0;i<num_cpus_cores;i++){
		d_g_state->panda_cpu_task_info[i].d_g_state = d_g_state;
		d_g_state->panda_cpu_task_info[i].cpu_job_conf = job_conf;
		d_g_state->panda_cpu_task_info[i].num_cpus_cores = num_cpus_cores;
		d_g_state->panda_cpu_task_info[i].start_idx = 0;
		d_g_state->panda_cpu_task_info[i].end_idx = 0;
	}//for
	d_g_state->configured = true;
	DoLog("CPU_GROUP_ID:[%d] DONE",d_g_state->cpu_group_id);

}


//For Version 0.2 depressed
void InitCPUMapReduce(cpu_context* d_g_state)
{	
#ifdef ABC
	if (d_g_state->num_input_record<=0) { DoLog("Error: no any input keys"); exit(-1);}
	if (d_g_state->input_keyval_arr == NULL) { DoLog("Error: input_keyval_arr == NULL"); exit(-1);}
	if (d_g_state->num_cpus_cores <= 0) {	DoLog("Error: d_g_state->num_cpus == 0"); exit(-1);}

	//DoLog("d_g_state->configured:%s  enable for iterative applications",d_g_state->configured? "true" : "false");
	//if (d_g_state->configured)
	//	return;

	DoLog("d_g_state->num_input_record:%d",d_g_state->num_input_record);
	int totalKeySize = 0;
	int totalValSize = 0;

	for(int i=0;i<d_g_state->num_input_record;i++){
		totalKeySize += d_g_state->input_keyval_arr[i].keySize;
		totalValSize += d_g_state->input_keyval_arr[i].valSize;
	}//for
	DoLog("totalKeySize:%d totalValSize:%d num_cpus:%d", totalKeySize, totalValSize, d_g_state->num_cpus_cores);
		
	//TODO determin num_cpus
	
	//d_g_state->num_cpus = 12;

	int num_cpus_cores = d_g_state->num_cpus_cores;
	d_g_state->panda_cpu_task = (pthread_t *)malloc(sizeof(pthread_t)*(num_cpus_cores));
	d_g_state->panda_cpu_task_info = (panda_cpu_task_info_t *)malloc(sizeof(panda_cpu_task_info_t)*(num_cpus_cores));

	d_g_state->intermediate_keyval_arr_arr_p = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*d_g_state->num_input_record);
	memset(d_g_state->intermediate_keyval_arr_arr_p, 0, sizeof(keyval_arr_t)*d_g_state->num_input_record);

	for (int i=0;i<num_cpus_cores;i++){
		d_g_state->panda_cpu_task_info[i].d_g_state = d_g_state;
		d_g_state->panda_cpu_task_info[i].num_cpus_cores = num_cpus_cores;
		d_g_state->panda_cpu_task_info[i].start_idx = 0;
		d_g_state->panda_cpu_task_info[i].end_idx = 0;
	}//for
	d_g_state->configured = true;
	DoLog("DONE");
#endif
}//void
	
void InitGPUMapReduce4(thread_info_t* thread_info)
{	

	gpu_context *d_g_state = (gpu_context *)(thread_info->d_g_state);
	job_configuration* gpu_job_conf = (job_configuration*)(thread_info->job_conf);
	keyval_t * kv_p = gpu_job_conf->input_keyval_arr;

	DoLog("d_g_state->configured:%s  enable for iterative applications",d_g_state->configured? "true" : "false");
	//if (d_g_state->configured)
	//	return;

	DoLog("copy %d input records from Host to GPU memory",gpu_job_conf->num_input_record);
	//checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_arr,sizeof(keyval_t)*d_g_state->num_input_record));
	int totalKeySize = 0;
	int totalValSize = 0;
	for(int i=0;i<gpu_job_conf->num_input_record;i++){
		totalKeySize += kv_p[i].keySize;
		totalValSize += kv_p[i].valSize;
	}//for
	DoLog("totalKeySize:%d totalValSize:%d", totalKeySize, totalValSize);
	
	void *input_vals_shared_buff = malloc(totalValSize);
	void *input_keys_shared_buff = malloc(totalKeySize);
	keyval_pos_t *input_keyval_pos_arr = (keyval_pos_t *)malloc(sizeof(keyval_pos_t)*gpu_job_conf->num_input_record);
	
	int keyPos = 0;
	int valPos = 0;
	int keySize = 0;
	int valSize = 0;
	
	for(int i=0; i<gpu_job_conf->num_input_record; i++){
		
		keySize = kv_p[i].keySize;
		valSize = kv_p[i].valSize;
		
		memcpy((char *)input_keys_shared_buff + keyPos,(char *)(kv_p[i].key), keySize);
		memcpy((char *)input_vals_shared_buff + valPos,(char *)(kv_p[i].val), valSize);
		
		input_keyval_pos_arr[i].keySize = keySize;
		input_keyval_pos_arr[i].keyPos = keyPos;
		input_keyval_pos_arr[i].valPos = valPos;
		input_keyval_pos_arr[i].valSize = valSize;

		keyPos += keySize;	
		valPos += valSize;
	}//for

	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_pos_arr,sizeof(keyval_pos_t)*gpu_job_conf->num_input_record));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keys_shared_buff, totalKeySize));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_vals_shared_buff, totalValSize));

	checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_pos_arr, input_keyval_pos_arr,sizeof(keyval_pos_t)*gpu_job_conf->num_input_record ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_keys_shared_buff, input_keys_shared_buff,totalKeySize ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_vals_shared_buff, input_vals_shared_buff,totalValSize ,hipMemcpyHostToDevice));

	//checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_arr,h_buff,sizeof(keyval_t)*d_g_state->num_input_record,hipMemcpyHostToDevice));
	hipDeviceSynchronize(); 
	d_g_state->configured = true;
	//(thread_info->d_g_state) = d_g_state;
	//printData2<<<1,1>>>(*d_g_state);
}//void

void InitGPUMapReduce3(gpu_context* d_g_state)
{	

	DoLog("d_g_state->configured:%s  enable for iterative applications",d_g_state->configured? "true" : "false");
	//if (d_g_state->configured)
	//	return;
	
	//checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_arr,sizeof(keyval_t)*d_g_state->num_input_record));
	int totalKeySize = 0;
	int totalValSize = 0;
	for(int i=0;i<d_g_state->num_input_record;i++){
		totalKeySize += d_g_state->h_input_keyval_arr[i].keySize;
		totalValSize += d_g_state->h_input_keyval_arr[i].valSize;
	}//for
	DoLog("copy %d input records from Host to GPU memory totalKeySize:%d totalValSize:%d",d_g_state->num_input_record, totalKeySize, totalValSize);
	
	void *input_vals_shared_buff = malloc(totalValSize);
	void *input_keys_shared_buff = malloc(totalKeySize);
	keyval_pos_t *input_keyval_pos_arr = (keyval_pos_t *)malloc(sizeof(keyval_pos_t)*d_g_state->num_input_record);
	
	int keyPos = 0;
	int valPos = 0;
	int keySize = 0;
	int valSize = 0;
	
	for(int i=0;i<d_g_state->num_input_record;i++){
		
		keySize = d_g_state->h_input_keyval_arr[i].keySize;
		valSize = d_g_state->h_input_keyval_arr[i].valSize;
		
		memcpy((char *)input_keys_shared_buff + keyPos,(char *)(d_g_state->h_input_keyval_arr[i].key), keySize);
		memcpy((char *)input_vals_shared_buff + valPos,(char *)(d_g_state->h_input_keyval_arr[i].val), valSize);
		
		input_keyval_pos_arr[i].keySize = keySize;
		input_keyval_pos_arr[i].keyPos = keyPos;
		input_keyval_pos_arr[i].valPos = valPos;
		input_keyval_pos_arr[i].valSize = valSize;

		keyPos += keySize;	
		valPos += valSize;
	}//for

	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_pos_arr,sizeof(keyval_pos_t)*d_g_state->num_input_record));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keys_shared_buff, totalKeySize));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_vals_shared_buff, totalValSize));

	checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_pos_arr, input_keyval_pos_arr,sizeof(keyval_pos_t)*d_g_state->num_input_record ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_keys_shared_buff, input_keys_shared_buff,totalKeySize ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_vals_shared_buff, input_vals_shared_buff,totalValSize ,hipMemcpyHostToDevice));

	//checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_arr,h_buff,sizeof(keyval_t)*d_g_state->num_input_record,hipMemcpyHostToDevice));
	hipDeviceSynchronize(); 
	d_g_state->configured = true;

	//printData2<<<1,1>>>(*d_g_state);

}//void

#if 0
void InitGPUMapReduce2(gpu_context* d_g_state)
{	
	
	DoLog("d_g_state->num_input_record:%d",d_g_state->num_input_record);
	//checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_arr,sizeof(keyval_t)*d_g_state->num_input_record));

	int totalKeySize = 0;
	int totalValSize = 0;

	for(int i=0;i<d_g_state->num_input_record;i++){
		totalKeySize += d_g_state->h_input_keyval_arr[i].keySize;
		totalValSize += d_g_state->h_input_keyval_arr[i].valSize;
	}//for

	void *input_vals_shared_buff = malloc(totalValSize);
	void *input_keys_shared_buff = malloc(totalKeySize);
	keyval_pos_t *input_keyval_pos_arr = (keyval_pos_t *)malloc(sizeof(keyval_pos_t)*d_g_state->num_input_record);

	int keyPos = 0;
	int valPos = 0;
	int keySize = 0;
	int valSize = 0;

	for(int i=0;i<d_g_state->num_input_record;i++){
		
		keySize = d_g_state->h_input_keyval_arr[i].keySize;
		valSize = d_g_state->h_input_keyval_arr[i].valSize;
		
		memcpy((char *)input_keys_shared_buff + keyPos,(char *)(d_g_state->h_input_keyval_arr[i].key), keySize);
		memcpy((char *)input_vals_shared_buff + valPos,(char *)(d_g_state->h_input_keyval_arr[i].val), valSize);
		
		input_keyval_pos_arr[i].keySize = keySize;
		input_keyval_pos_arr[i].keyPos = keyPos;
		input_keyval_pos_arr[i].valPos = valPos;
		input_keyval_pos_arr[i].valSize = valSize;

		keyPos += keySize;	
		valPos += valSize;

	}//for

	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keyval_pos_arr,sizeof(keyval_pos_t)*d_g_state->num_input_record));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_keys_shared_buff, totalKeySize));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_input_vals_shared_buff, totalValSize));

	checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_pos_arr, input_keyval_pos_arr,sizeof(keyval_pos_t)*d_g_state->num_input_record ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_keys_shared_buff, input_keys_shared_buff,totalKeySize ,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_input_vals_shared_buff, input_vals_shared_buff,totalValSize ,hipMemcpyHostToDevice));

	//checkCudaErrors(hipMemcpy(d_g_state->d_input_keyval_arr,h_buff,sizeof(keyval_t)*d_g_state->num_input_record,hipMemcpyHostToDevice));
	hipDeviceSynchronize(); 

}//void
#endif


/*
void InitMapReduce(Spec_t* spec)
{
	if (g_spec->dimBlockMap <= 0)
		g_spec->dimBlockMap = DEFAULT_DIMBLOCK;
	if (g_spec->dimBlockReduce <= 0)
		g_spec->dimBlockReduce = DEFAULT_DIMBLOCK;
	if (g_spec->numRecTaskReduce <= 0)
		g_spec->numRecTaskReduce = DEFAULT_NUMTASK;
	if (g_spec->numRecTaskMap <= 0)
		g_spec->numRecTaskMap = DEFAULT_NUMTASK;
	if (g_spec->workflow <= 0)
		g_spec->workflow = MAP_ONLY;
}*/

void InitCPUDevice(thread_info_t*thread_info){

	//------------------------------------------
	//1, init CPU device
	//------------------------------------------
	
	cpu_context *d_g_state = (cpu_context *)(thread_info->d_g_state);
	if (d_g_state->num_cpus_cores<=0) d_g_state->num_cpus_cores = getCPUCoresNum();

	int tid = thread_info->tid;
	DoLog( "Init CPU Deivce tid:%d",tid);
	//char *fn = thread_info->file_name;
	//"% num_gpus" allows more CPU threads than GPU devices
	
}

void InitGPUDevice(thread_info_t*thread_info){
	
	//------------------------------------------
	//1, init device
	//------------------------------------------
	
	gpu_context *d_g_state = (gpu_context *)(thread_info->d_g_state);
	int tid = thread_info->tid;
	int num_gpus = d_g_state->num_gpus;
	if (num_gpus == 0) {
		DoLog("error num_gpus == 0");
		exit(-1);
	}
	
	int gpu_id;
	hipGetDevice(&gpu_id);
	int gpu_count = 0;
	hipGetDeviceCount(&gpu_count);

	DoLog("check GPU Device IDs -> tid:%d current gpu_id:%d num_gpus by user:%d hipGetDeviceCount:%d", tid, gpu_id, num_gpus, gpu_count);

	if ( gpu_id != tid ){
		//DoLog("hipSetDevice gpu_id %d == (tid num_gpus) %d ", gpu_id, tid%num_gpus);
		hipSetDevice(tid % num_gpus);  
	}//if
	//DoLog("------------------------------------------------------------------------");

	hipGetDevice(&gpu_id);
	d_g_state->gpu_id = gpu_id;
		
	size_t total_mem,avail_mem, heap_limit;
	checkCudaErrors(hipMemGetInfo( &avail_mem, &total_mem ));
	hipDeviceSetLimit(hipLimitMallocHeapSize, (int)(total_mem*0.2)); 
	hipDeviceGetLimit(&heap_limit, hipLimitMallocHeapSize);

	DoLog("TID:[%d] num_gpus:%d gpu_id:%d device_name:%s",tid,num_gpus,gpu_id,thread_info->device_name);
	DoLog("GPU_ID:[%d] hipLimitMallocHeapSize:%d MB avail_mem:%d MB total_mem:%d MB",gpu_id, heap_limit/1024/1024, avail_mem/1024/1024,total_mem/1024/1024);

}

//Ratio = Tcpu/Tgpu
//Tcpu = (execution time on CPU cores for sampled tasks)/(#sampled tasks)
//Tgpu = (execution time on 1 GPU for sampled tasks)/(#sampled tasks)

float Smart_Scheduler(job_configuration *job_conf){
#ifdef ABC	
	int num_gpus = 1;//job_conf->num_gpus;
	int num_cpus_cores = getCPUCoresNum();//job_conf->num_cpus;
	if (num_cpus_cores >2)
		num_cpus_cores -= 2;
	int num_cpus_group = 1;//job_conf->num_cpus_groups;
	panda_context *panda = GetPandaContext();
	
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_group));
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*(num_gpus + num_cpus_group));
	
	for (int i=0; i<num_gpus; i++){
		thread_info[i].tid = i;
		//thread_info[i].file_name = argv[i+1];
		thread_info[i].num_gpus = num_gpus;
		thread_info[i].device_type = GPU_ACC;
		
		hipDeviceProp_t gpu_dev;
		hipGetDeviceProperties(&gpu_dev, i);
	
		thread_info[i].device_name = gpu_dev.name;
		gpu_context *d_g_state = GetGPUContext();
		//d_g_state->matrix_size = job_conf->matrix_size;
		d_g_state->num_mappers = job_conf->num_mappers;
		d_g_state->num_reducers = job_conf->num_reducers;
		thread_info[i].d_g_state = d_g_state;
	}//for num_gpus
	
	for (int i=num_gpus; i<num_gpus+num_cpus_group; i++){
		thread_info[i].tid = i;
		thread_info[i].device_type = CPU_ACC;
		cpu_context *d_g_state = GetCPUContext();
		d_g_state->num_cpus_cores = num_cpus_cores;
		thread_info[i].d_g_state = d_g_state;
	}//for
	
	DoLog("num_gpus:%d num_cpus_group:%d num_input_record:%d sizeof(int):%d\n", num_gpus, num_cpus_group,job_conf->num_input_record,sizeof(int));

	int cpu_sampled_tasks_num = 0;
	int gpu_sampled_tasks_num = 0;
	cpu_sampled_tasks_num = num_cpus_cores*job_conf->auto_tuning_sample_rate;
	gpu_sampled_tasks_num = getGPUCoresNum()*job_conf->auto_tuning_sample_rate;

	if (cpu_sampled_tasks_num>job_conf->num_input_record)
		cpu_sampled_tasks_num = job_conf->num_input_record;

	if (gpu_sampled_tasks_num>job_conf->num_input_record)
		gpu_sampled_tasks_num = job_conf->num_input_record;

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_group); dev_id++){
		int start_id = 0;
		int end_id = 0;//job_conf->num_cpus_cores*2; //(job_conf->num_input_record/100);
		
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				end_id = start_id + gpu_sampled_tasks_num;
				AddMapInputRecordGPU(d_g_state,(job_conf->input_keyval_arr), start_id, end_id);
			}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				end_id = start_id + cpu_sampled_tasks_num;
				AddMapInputRecordCPU(d_g_state,(job_conf->input_keyval_arr), start_id, end_id);
				
		}//if
	}//for

	double t1 = PandaTimer();
	Panda_Map((void *)&(thread_info[0]));
	double t2 = PandaTimer();
	//start cpu 
	Panda_Map((void *)&(thread_info[1]));
	double t3 = PandaTimer();
	
	double t_cpu = (t3-t2);///cpu_sampled_tasks_num;
	double t_gpu = (t2-t1);///gpu_sampled_tasks_num;

	if (t_gpu<0.0001)
		t_gpu=0.0001;

	double ratio = (t_cpu*gpu_sampled_tasks_num)/(t_gpu*cpu_sampled_tasks_num);
	char log[128];
	sprintf(log,"	cpu_sampled_tasks:%d cpu time:%f cpu time per task:%f", cpu_sampled_tasks_num, t_cpu, t_cpu/(cpu_sampled_tasks_num));
	DoDiskLog(log);
	sprintf(log,"	gpu_sampled_tasks:%d gpu time:%f gpu time per task:%f	ratio:%f", gpu_sampled_tasks_num, t_gpu, t_gpu/(gpu_sampled_tasks_num), ratio);
	DoDiskLog(log);
	#endif

	return (1.0);


}//void

//--------------------------------------------------
//  Panda_Job_Scheduler
//--------------------------------------------------
/*
 * 1) input a set of panda worker (thread)
 * 2) each panda worker consist of one panda job and pand device
 * 3) copy input data from pand job to pand device
 *
 */
void PandaMetaScheduler(thread_info_t *thread_info, int num_gpus, int num_cpus_groups){

	panda_context *panda = GetPandaContext();
	panda->num_gpus = num_gpus;
	panda->num_cpus_groups = num_cpus_groups;
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_groups));
	
	for (int dev_id=0; dev_id<(num_gpus + num_cpus_groups); dev_id++){

		if (thread_info[dev_id].device_type == GPU_ACC){
			
			job_configuration* gpu_job_conf = (job_configuration*)(thread_info[dev_id].job_conf);
			gpu_context *d_g_state = GetGPUContext();
			d_g_state->num_mappers = gpu_job_conf->num_mappers;
			d_g_state->num_reducers = gpu_job_conf->num_reducers;
			d_g_state->num_gpus = num_gpus;
			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;
			DoLog("DEV_ID:[%d] GPU_ACC TID:%d",dev_id,thread_info[dev_id].tid);
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){

			cpu_context *d_g_state = GetCPUContext();
			thread_info[dev_id].tid = dev_id;
			thread_info[dev_id].d_g_state = d_g_state;
			DoLog("DEV_ID:[%d] CPU_ACC TID:%d",dev_id,thread_info[dev_id].tid);

		}//if
	}//for

	///////////////////////////////////////////////////
	
	//DoLog("num_cpus_group:%d num_cpu_input_record:", num_cpus_groups /*,cpu_job_conf->num_input_record*/);
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){

		if (thread_info[dev_id].device_type == GPU_ACC){

				/*int num_input_record_per_gpu = gpu_job_conf->num_input_record/num_gpus;
				int start_id = num_input_record_per_gpu*dev_id;
				int end_id = start_id + num_input_record_per_gpu;
				DoLog("num_gpus:%d num_gpu_input_record:%d start:%d end:%d", num_gpus , gpu_job_conf->num_input_record,start_id,end_id);
				if (dev_id == num_gpus -1)
					end_id = gpu_job_conf->num_input_record;*/
				job_configuration *gpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_id = 0;
				int end_id = gpu_job_conf->num_input_record;
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);

				AddMapInputRecordGPU(d_g_state,(gpu_job_conf->input_keyval_arr), start_id,end_id);
				
		}//if
	
		if (thread_info[dev_id].device_type == CPU_ACC){

				job_configuration *cpu_job_conf = (job_configuration *)(thread_info[dev_id].job_conf);
				int start_id = 0;
				int end_id = cpu_job_conf->num_input_record;
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				
				AddMapInputRecordCPU(d_g_state,(cpu_job_conf->input_keyval_arr),start_id, end_id);
				
		}//if
	}//for
	
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
		//if (thread_info[dev_id].device_type == GPU_ACC){
		if (pthread_create(&(no_threads[dev_id]), NULL, Panda_Map, (char *)&(thread_info[dev_id])) != 0) 
			perror("Thread creation failed!\n");
	}//for

	for (int i=0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for

	//DoLog("start to merge for GPU's and CPU's results");
	for (int i = 0; i < num_gpus+num_cpus_groups; i++){
		if (thread_info[i].device_type == CPU_ACC)
			Panda_Shuffle_Merge_CPU((panda_context*)panda, (cpu_context*)(thread_info[i].d_g_state));
		if (thread_info[i].device_type == GPU_ACC)
			Panda_Shuffle_Merge_GPU((panda_context*)panda, (gpu_context*)(thread_info[i].d_g_state));
	}//for
	//DoLog("totoal number of different intermediate records:%d",panda->sorted_keyvals_arr_len);

	int num_sorted_intermediate_record = panda->sorted_keyvals_arr_len;
	int records_per_device = num_sorted_intermediate_record/(num_gpus*10+num_cpus_groups);
	int *split = (int*)malloc(sizeof(int)*(num_gpus+num_cpus_groups));
	for (int i=0;i<num_gpus;i++){
			split[i] = records_per_device*10*(i+1);
	}//for
	for (int i=num_gpus;i<num_gpus+num_cpus_groups;i++){
			split[i] = records_per_device*10*(num_gpus)+(i+1)*records_per_device;
	}//for
	split[num_gpus + num_cpus_groups-1] = num_sorted_intermediate_record;

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
		int start_id = 0;
		if (dev_id>0) start_id = split[dev_id-1];
		int end_id = split[dev_id];
				
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id; i<end_id; i++){
				AddReduceInputRecordGPU(d_g_state,(panda->sorted_intermediate_keyvals_arr),start_id, end_id);
				//}//for

		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id;i<end_id;i++){
				AddReduceInputRecordCPU(d_g_state,(panda->sorted_intermediate_keyvals_arr),start_id, end_id);
				//}//for
		}//if
	}//for

	for (int dev_id = 0; dev_id < (num_gpus+num_cpus_groups); dev_id++){
		if (pthread_create(&(no_threads[dev_id]),NULL,Panda_Reduce,(char *)&(thread_info[dev_id]))!=0) 
			perror("Thread creation failed!\n");
	}//for
		
	for (int i=0; i < num_gpus + num_cpus_groups; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	//Panda_Reduce(&thread_info[num_gpus-1]);															 //
	///////////////////////////////////////////////////////////////////////////////////////////////////////

	DoLog("Finishing Panda MapReduce Job...");
	int total_output_records = 0;
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_groups); dev_id++){
	
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->d_reduced_keyval_arr_len;
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->sorted_keyvals_arr_len;
		}//if
	}//for
	DoLog("there are :%d output records\n",total_output_records);
	DoLog("=====finish map/reduce=====");

}//PandaMetaScheduler



void Start_Panda_Job(job_configuration *job_conf){
#ifdef ABC	
	int num_gpus = job_conf->num_gpus;
	int num_cpus_cores = job_conf->num_cpus_cores;
	int num_cpus_group = job_conf->num_cpus_groups;

	panda_context *panda = GetPandaContext();
	
	panda->num_gpus = num_gpus;
	panda->num_cpus_groups = num_cpus_group;
	DoLog("Start num_gpus:%d  num_cpus_groups:%d", num_gpus, num_cpus_group);
	pthread_t *no_threads = (pthread_t*)malloc(sizeof(pthread_t)*(num_gpus + num_cpus_group));
	thread_info_t *thread_info = (thread_info_t*)malloc(sizeof(thread_info_t)*(num_gpus + num_cpus_group));
	
	for (int i=0; i<num_gpus; i++){
		thread_info[i].tid = i;
		//thread_info[i].file_name = argv[i+1];
		thread_info[i].num_gpus = num_gpus;
		thread_info[i].device_type = GPU_ACC;
		
		hipDeviceProp_t gpu_dev;
		hipGetDeviceProperties(&gpu_dev, i);
		
		//DoLog("Configure Device ID:%d: Device Name:%s MultProcessorCount:%d sm_per_multiproc:%d", i, gpu_dev.name,gpu_dev.multiProcessorCount,sm_per_multiproc);

		thread_info[i].device_name = gpu_dev.name;
		gpu_context *d_g_state = GetGPUContext();
		d_g_state->matrix_size = job_conf->matrix_size;
		d_g_state->num_mappers = job_conf->num_mappers;
		d_g_state->num_reducers = job_conf->num_reducers;
		thread_info[i].d_g_state = d_g_state;
	}//for num_gpus
	
	
	for (int i=num_gpus; i<num_gpus+num_cpus_group; i++){
		thread_info[i].tid = i;
		thread_info[i].device_type = CPU_ACC;
		cpu_context *d_g_state = GetCPUContext();
		d_g_state->num_cpus_cores = num_cpus_cores;
		thread_info[i].d_g_state = d_g_state;
	}//for


	///////////////////////////////////////////////////
	double ratio = 10.0;
	ratio = (double)(job_conf->ratio);
	if (job_conf->auto_tuning){
		ratio = (Smart_Scheduler(job_conf));
		job_conf->ratio = ratio;
	}//if
	
	//////////////////////////////////////////////////
	DoLog("num_gpus:%d num_cpus_group:%d num_input_record:%d sizeof(int):%d  ratio:%f\n", num_gpus, num_cpus_group,job_conf->num_input_record,sizeof(int),ratio);
	//DoLog("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! Smart Scheduler Test");	return;

	int *split = NULL;
	split = (int *)malloc(sizeof(int)*(num_gpus+num_cpus_group));
	
	int num_input_record = job_conf->num_input_record;
	int records_per_device = (int)(num_input_record/(num_gpus*ratio+num_cpus_group));
	for (int i=0;i<num_gpus;i++){
			split[i] = (int)(records_per_device*ratio*(i+1));
	}//for

	for (int i=num_gpus;i<num_gpus+num_cpus_group;i++){
			split[i] = (int)(records_per_device*ratio*(num_gpus)+(i+1)*records_per_device);
	}//for
	split[num_gpus+num_cpus_group-1] = num_input_record;

	printf("---	split:num_input_record:%d records_per_device:%d  ",num_input_record, records_per_device);
	for (int i=0;i<num_gpus+num_cpus_group;i++)
		printf("%d\t",split[i]);
	printf("\n");

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_group); dev_id++){
		int start_id = 0;
		if (dev_id>0) start_id = split[dev_id-1];

		int end_id = split[dev_id];
		
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id; i<end_id; i++){
				//printf(":%s  keySize:%d",job_conf->input_keyval_arr[i].val, job_conf->input_keyval_arr[i].valSize);
				AddMapInputRecordGPU(d_g_state,(job_conf->input_keyval_arr), start_id,end_id);
				//}//for
			}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id;i<end_id;i++){
				AddMapInputRecordCPU(d_g_state,(job_conf->input_keyval_arr),start_id, end_id);
				//}//for
		}//if
	}//for

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_group); dev_id++){
		//if (thread_info[dev_id].device_type == GPU_ACC){
		if (pthread_create(&(no_threads[dev_id]), NULL, Panda_Map, (char *)&(thread_info[dev_id])) != 0) 
			perror("Thread creation failed!\n");
	}//for

	for (int i=0; i < num_gpus + num_cpus_group; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for

	DoLog("start to merge!");
	for (int i = 0; i<num_gpus; i++){
		Panda_Shuffle_Merge_GPU((panda_context*)panda, (gpu_context*)(thread_info[i].d_g_state));
	}//for

	for (int i = num_gpus; i < num_gpus+num_cpus_group; i++){
		Panda_Shuffle_Merge_CPU((panda_context*)panda, (cpu_context*)(thread_info[i].d_g_state));
	}//for

	DoLog("totoal number of different intermediate records:%d",panda->sorted_keyvals_arr_len);
	//TOD smart job for reduce ratio

	//hipDeviceSynchronize();
	//static scheduling -- split the workload between devices 
	int num_sorted_intermediate_record = panda->sorted_keyvals_arr_len;
	records_per_device = num_sorted_intermediate_record/(num_gpus*10+num_cpus_group);
	for (int i=0;i<num_gpus;i++){
			split[i] = records_per_device*10*(i+1);
	}//for
	for (int i=num_gpus;i<num_gpus+num_cpus_group;i++){
			split[i] = records_per_device*10*(num_gpus)+(i+1)*records_per_device;
	}//for
	split[num_gpus + num_cpus_group-1] = num_sorted_intermediate_record;

	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_group); dev_id++){
		int start_id = 0;
		if (dev_id>0) start_id = split[dev_id-1];
		int end_id = split[dev_id];
				
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id; i<end_id; i++){
				AddReduceInputRecordGPU(d_g_state,(panda->sorted_intermediate_keyvals_arr),start_id, end_id);
				//}//for

		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				//for (int i=start_id;i<end_id;i++){
				AddReduceInputRecordCPU(d_g_state,(panda->sorted_intermediate_keyvals_arr),start_id, end_id);
				//}//for
		}//if
		

	}//for

	for (int dev_id = 0; dev_id < (num_gpus+num_cpus_group); dev_id++){
		if (pthread_create(&(no_threads[dev_id]),NULL,Panda_Reduce,(char *)&(thread_info[dev_id]))!=0) 
			perror("Thread creation failed!\n");
	}//for
		
	for (int i=0; i < num_gpus + num_cpus_group; i++){
		void *exitstat;
		if (pthread_join(no_threads[i],&exitstat)!=0) perror("joining failed");
	}//for
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	//Panda_Reduce(&thread_info[num_gpus-1]);
	///////////////////////////////////////////////////////////////////////////////////////////////////////

	DoLog("Finishing Panda MapReduce Job...");
	int total_output_records = 0;
	for (int dev_id = 0; dev_id<(num_gpus+num_cpus_group); dev_id++){
	
		if (thread_info[dev_id].device_type == GPU_ACC){
				gpu_context* d_g_state = (gpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->d_reduced_keyval_arr_len;
		}//if

		if (thread_info[dev_id].device_type == CPU_ACC){
				cpu_context* d_g_state = (cpu_context*)(thread_info[dev_id].d_g_state);
				total_output_records += d_g_state->sorted_keyvals_arr_len;
		}//if
	}//for
	DoLog("there are :%d output records\n",total_output_records);
	DoLog("=====finish map/reduce=====");
#endif
}		

void AddPandaTask(job_configuration* job_conf,
						void*		key, 
						void*		val,
						int		keySize, 
						int		valSize){
	
	int len = job_conf->num_input_record;
	if (len<0) return;
	if (len == 0) job_conf->input_keyval_arr = NULL;

	job_conf->input_keyval_arr = (keyval_t *)realloc(job_conf->input_keyval_arr, sizeof(keyval_t)*(len+1));
	job_conf->input_keyval_arr[len].keySize = keySize;
	job_conf->input_keyval_arr[len].valSize = valSize;
	job_conf->input_keyval_arr[len].key = malloc(keySize);
	job_conf->input_keyval_arr[len].val = malloc(valSize);

	memcpy(job_conf->input_keyval_arr[len].key,key,keySize);
	memcpy(job_conf->input_keyval_arr[len].val,val,valSize);
	job_conf->num_input_record++;
	
}

void AddReduceInputRecordGPU(gpu_context* d_g_state, keyvals_t * sorted_intermediate_keyvals_arr, int starti, int endi){
	
	long total_count = 0;
	for(int i=starti;i<endi;i++){
		total_count += sorted_intermediate_keyvals_arr[i].val_arr_len;
	}//for
	
	int totalKeySize = 0;
	int totalValSize = 0;
	for(int i=starti;i<endi;i++){
		totalKeySize += sorted_intermediate_keyvals_arr[i].keySize;
		for (int j=0;j<sorted_intermediate_keyvals_arr[i].val_arr_len;j++)
		totalValSize += sorted_intermediate_keyvals_arr[i].vals[j].valSize;
	}//for
	DoLog("totalKeySize:%d totalValSize:%d ",totalKeySize,totalValSize);
		
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_sorted_keys_shared_buff,totalKeySize));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_sorted_vals_shared_buff,totalValSize));
	checkCudaErrors(hipMalloc((void **)&d_g_state->d_keyval_pos_arr,sizeof(keyval_pos_t)*total_count));
	
	d_g_state->h_sorted_keys_shared_buff = malloc(sizeof(char)*totalKeySize);
	d_g_state->h_sorted_vals_shared_buff = malloc(sizeof(char)*totalValSize);
	
	char *sorted_keys_shared_buff = (char *)d_g_state->h_sorted_keys_shared_buff;
	char *sorted_vals_shared_buff = (char *)d_g_state->h_sorted_vals_shared_buff;
	char *keyval_pos_arr = (char *)malloc(sizeof(keyval_pos_t)*total_count);
	
	int sorted_key_arr_len = (endi-starti);
	keyval_pos_t *tmp_keyval_pos_arr = (keyval_pos_t *)malloc(sizeof(keyval_pos_t)*total_count);
	DoLog("total number of different intermediate records:%d total records:%d", endi-starti, total_count);
	int *pos_arr_4_pos_arr = (int*)malloc(sizeof(int)*(sorted_key_arr_len));
	memset(pos_arr_4_pos_arr,0,sizeof(int)*sorted_key_arr_len);

	int index = 0;
	int keyPos = 0;
	int valPos = 0;
	for (int i=starti;i<endi;i++){
		keyvals_t* p = (keyvals_t*)&(sorted_intermediate_keyvals_arr[i]);
		memcpy(sorted_keys_shared_buff+keyPos,p->key, p->keySize);
		
		for (int j=0;j<p->val_arr_len;j++){
			tmp_keyval_pos_arr[index].keyPos = keyPos;
			tmp_keyval_pos_arr[index].keySize = p->keySize;
			tmp_keyval_pos_arr[index].valPos = valPos;
			tmp_keyval_pos_arr[index].valSize = p->vals[j].valSize;
			memcpy(sorted_vals_shared_buff + valPos,p->vals[j].val,p->vals[j].valSize);
			valPos += p->vals[j].valSize;
			index++;
		}//for
		keyPos += p->keySize;
		pos_arr_4_pos_arr[i-starti] = index;
	}//

	d_g_state->d_sorted_keyvals_arr_len = endi-starti;
	checkCudaErrors(hipMemcpy(d_g_state->d_keyval_pos_arr,tmp_keyval_pos_arr,sizeof(keyval_pos_t)*total_count,hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_g_state->d_pos_arr_4_sorted_keyval_pos_arr,sizeof(int)*sorted_key_arr_len));
	checkCudaErrors(hipMemcpy(d_g_state->d_pos_arr_4_sorted_keyval_pos_arr,pos_arr_4_pos_arr,sizeof(int)*sorted_key_arr_len,hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_g_state->d_sorted_keys_shared_buff, sorted_keys_shared_buff, sizeof(char)*totalKeySize,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_g_state->d_sorted_vals_shared_buff, sorted_vals_shared_buff, sizeof(char)*totalValSize,hipMemcpyHostToDevice));

}


void AddMapInputRecordGPU(gpu_context* d_g_state,
						keyval_t *kv_p, int start_id, int end_id){
		
	int len = d_g_state->num_input_record;
	//DoLog("len:%d realloc:%d",len,sizeof(keyval_t)*(len+1));
	if (len == 0) d_g_state->h_input_keyval_arr = NULL;
	if (len<0) return;
	if (end_id<=start_id) return;
	DoLog("add map input record for gpu context current len:%d added len:%d", len, end_id - start_id);			

	d_g_state->h_input_keyval_arr = (keyval_t *)realloc(d_g_state->h_input_keyval_arr, sizeof(keyval_t)*(len+end_id - start_id));
	//assert(d_g_state->h_input_keyval_arr != NULL);
	for (int i=start_id;i<end_id;i++){
	d_g_state->h_input_keyval_arr[len].keySize = kv_p[i].keySize;
	d_g_state->h_input_keyval_arr[len].valSize = kv_p[i].valSize;
	d_g_state->h_input_keyval_arr[len].key = kv_p[i].key;
	d_g_state->h_input_keyval_arr[len].val = kv_p[i].val;
	//memcpy(d_g_state->h_input_keyval_arr[len].key,key,keySize);
	//memcpy(d_g_state->h_input_keyval_arr[len].val,val,valSize);
	d_g_state->num_input_record++;
	len++;
	}
	//DoLog("added %d map tasks",end_id-start_id);
			
}


void AddMapInputRecordCPU(cpu_context* d_g_state,
						keyval_t *kv_p, int starti, int endi){

	int len = d_g_state->num_input_record;
	if (len<0) return;
	if (len == 0) d_g_state->input_keyval_arr = NULL;
	DoLog("add map input record for cpu context current len:%d added len:%d",len,endi-starti);			
	//DoLog("len:%d size:%d",len,sizeof(keyval_t)*(len+1));
	d_g_state->input_keyval_arr = (keyval_t *)realloc(d_g_state->input_keyval_arr, sizeof(keyval_t)*(len+endi-starti));

	for (int i=starti;i<endi;i++){
	
	d_g_state->input_keyval_arr[len].keySize = kv_p[i].keySize;
	d_g_state->input_keyval_arr[len].valSize = kv_p[i].valSize;
	d_g_state->input_keyval_arr[len].key = kv_p[i].key;
	d_g_state->input_keyval_arr[len].val = kv_p[i].val;
	
	d_g_state->num_input_record++;
	len++;
	}

	//DoLog("add map input record len:%d",len);			
}

void AddReduceInputRecordCPU(cpu_context* d_g_state,
						keyvals_t *kv_p, int start_id, int end_id){
							
    if (end_id<start_id){	DoLog("error ! end_id <= start_id");		end_id = start_id;	}
							
	int len = d_g_state->sorted_keyvals_arr_len;
	if (len<0) {	DoLog("error ! len<0");		return;	}
	if (len == 0) d_g_state->sorted_intermediate_keyvals_arr = NULL;
	
	DoLog("start_id:%d, end_id:%d, len:%d\n",start_id,end_id,len);

	d_g_state->sorted_intermediate_keyvals_arr = (keyvals_t *)realloc(d_g_state->sorted_intermediate_keyvals_arr, 
		sizeof(keyvals_t)*(len+end_id-start_id));

	for (int i = len; i< len+end_id-start_id; i++){
		int test = kv_p[start_id+i-len].keySize;
		d_g_state->sorted_intermediate_keyvals_arr[i].keySize = kv_p[start_id+i-len].keySize;
		d_g_state->sorted_intermediate_keyvals_arr[i].key = kv_p[start_id+i-len].key;
		d_g_state->sorted_intermediate_keyvals_arr[i].vals = kv_p[start_id+i-len].vals;
		d_g_state->sorted_intermediate_keyvals_arr[i].val_arr_len = kv_p[start_id+i-len].val_arr_len;
		d_g_state->sorted_keyvals_arr_len++;
	}//for

}


__device__ void Emit2  (void*		key, 
						void*		val, 
						int		keySize, 
						int		valSize,
						gpu_context *d_g_state){
						
			keyval_t *p = &(d_g_state->d_reduced_keyval_arr[TID]);
			p->keySize = keySize;
			p->key = malloc(keySize);
			memcpy(p->key,key,keySize);
			p->valSize = valSize;
			p->val = malloc(valSize);
			memcpy(p->val,val,valSize);
			printf("[output]: key:%s  val:%d\n",key,*(int *)val);
						
}//__device__ 


void CPUEmitIntermediate(void *key, void *val, int keySize, int valSize, cpu_context *d_g_state, int map_task_idx){
	
	//printf(":%s   :%d\n",key, *(int*)val);
	keyval_arr_t *kv_arr_p = &(d_g_state->intermediate_keyval_arr_arr_p[map_task_idx]);
	//keyval_t *p = (keyval_t *)kv_arr_p->arr;
	//void *buff = kv_arr_p->buff;
	//&(kv_arr_p->arr[len]) = (keyval_t*)((char *)buff - sizeof(keyval_t));
	//keyval_t *kv_p = (keyval_t *)((char *)buff + kv_arr_p->buff_len - sizeof(keyval_t)*((*kv_arr_p->total_arr_len)+1));

	if (kv_arr_p->arr_len==0)
		kv_arr_p->arr = NULL;

	kv_arr_p->arr = (keyval_t*)realloc(kv_arr_p->arr, sizeof(keyval_t)*(kv_arr_p->arr_len+1));
	//(*kv_arr_p->total_arr_len)++;

	/*
	if (!(kv_arr_p->buff_pos +keySize+valSize < kv_arr_p->buff_len - sizeof(keyval_t)*((*kv_arr_p->total_arr_len)+1))){
		printf("!!!!!!!error there is not engough shared memory\n");
		return;
	}*/

	//printf("remain buff:%d\n", kv_arr_p->buff_len - sizeof(keyval_t)*(kv_arr_p->arr_len+1) - kv_arr_p->buff_pos);
	int current_map_output_index = (kv_arr_p->arr_len);
	keyval_t *kv_p = &(kv_arr_p->arr[current_map_output_index]);
	kv_p->key = (char *)malloc(sizeof(keySize));
	//kv_arr_p->buff_pos += keySize;
	memcpy(kv_p->key,key,keySize);
	kv_p->keySize = keySize;
	
	kv_p->val = (char *)malloc(sizeof(valSize));
	//kv_arr_p->buff_pos += valSize;
	memcpy(kv_p->val,val,valSize);
	kv_p->valSize = valSize;

	
	kv_arr_p->arr_len++;
	//DoLog("current d_g_state->intermediate_keyval_arr_arr_p len:%d count:%d",kv_arr_p->arr_len,count);
	//d_g_state->d_intermediate_keyval_total_count[map_task_idx] = kv_arr_p->arr_len;
	/*
	kv_p->key = (char *)(buff)+kv_arr_p->buff_pos;
	kv_arr_p->buff_pos += keySize;
	memcpy(kv_p->key,key,keySize);
	kv_p->keySize = keySize;
	
	kv_p->val = (char *)(buff)+kv_arr_p->buff_pos;
	kv_arr_p->buff_pos += valSize;
	memcpy(kv_p->val,val,valSize);
	kv_p->valSize = valSize;

	kv_arr_p->arr_len++;
	d_g_state->d_intermediate_keyval_total_count[map_task_idx] = kv_arr_p->arr_len;
	*/

	//printf("CPU EmitInterMediate2 map_task_id:%d, key:%s: keyval_arr_len:%d\n", map_task_idx, kv_p->key, kv_arr_p->arr_len);

}//__device__


__device__ void EmitIntermediate2(void *key, void *val, int keySize, int valSize, gpu_context *d_g_state, int map_task_idx){
	
	//printf(":%s   :%d\n",key, *(int*)val);
	keyval_arr_t *kv_arr_p = d_g_state->d_intermediate_keyval_arr_arr_p[map_task_idx];
	//keyval_t *p = (keyval_t *)kv_arr_p->arr;
	void *buff = kv_arr_p->buff;
	//&(kv_arr_p->arr[len]) = (keyval_t*)((char *)buff - sizeof(keyval_t));
	
	keyval_t *kv_p = (keyval_t *)((char *)buff + kv_arr_p->buff_len - sizeof(keyval_t)*((*kv_arr_p->total_arr_len)+1));
	kv_arr_p->arr = kv_p;
	(*kv_arr_p->total_arr_len)++;

	//TODO Hui Li 8/6/2012
	if (!(kv_arr_p->buff_pos +keySize+valSize < kv_arr_p->buff_len - sizeof(keyval_t)*((*kv_arr_p->total_arr_len)+1))){
		printf("!!!!!!!error there is not engough shared memory\n");
		return;
	}//
	//printf("remain buff:%d\n", kv_arr_p->buff_len - sizeof(keyval_t)*(kv_arr_p->arr_len+1) - kv_arr_p->buff_pos);
	//int current_map_output_index = (kv_arr_p->arr_len);
	//keyval_t *kv_p = &(kv_arr_p->arr[current_map_output_index]);

	kv_p->key = (char *)(buff)+kv_arr_p->buff_pos;
	kv_arr_p->buff_pos += keySize;
	memcpy(kv_p->key,key,keySize);
	kv_p->keySize = keySize;
	
	kv_p->val = (char *)(buff)+kv_arr_p->buff_pos;
	kv_arr_p->buff_pos += valSize;
	memcpy(kv_p->val,val,valSize);
	kv_p->valSize = valSize;

	kv_arr_p->arr_len++;
	d_g_state->d_intermediate_keyval_total_count[map_task_idx] = kv_arr_p->arr_len;
	//printf("EmitInterMediate2 TID[%d] map_task_id:%d, key%s: keyval_arr_len:%d\n",TID, map_task_idx, kv_p->key, kv_arr_p->arr_len);
}//__device__

//-------------------------------------------------
//called by user defined map function
//-------------------------------------------------

__global__ void GPUMapPartitioner(gpu_context d_g_state)
{	
	/*int index = TID;
	int bid = BLOCK_ID;
	int tid = THREAD_ID;*/

	int num_records_per_thread = (d_g_state.num_input_record+(gridDim.x*blockDim.x)-1)/(gridDim.x*blockDim.x);
	int block_start_idx = num_records_per_thread*blockIdx.x*blockDim.x;
	int thread_start_idx = block_start_idx 
		+ (threadIdx.x/STRIDE)*num_records_per_thread*STRIDE
		+ (threadIdx.x%STRIDE);
	int thread_end_idx = thread_start_idx+num_records_per_thread*STRIDE;
			
	if (thread_end_idx>d_g_state.num_input_record)
		thread_end_idx = d_g_state.num_input_record;

	if (thread_start_idx >= thread_end_idx)
		return;

	char * buff = (char *)malloc(sizeof(char)*1024*100);
	int * total_arr_len = (int*)malloc(sizeof(int));
	(*total_arr_len) = 0;

	keyval_arr_t *kv_arr_t_arr = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*(thread_end_idx-thread_start_idx+STRIDE-1)/STRIDE);
	int index = 0;

	//printf("Mapper TID:%d, thread_start_idx:%d  thread_end_idx:%d runed tasks:%d totalTasks:%d\n",
	//	TID, thread_start_idx,thread_end_idx,(thread_end_idx - thread_start_idx)/STRIDE,d_g_state.num_input_record);

	for(int map_task_idx=thread_start_idx; map_task_idx < thread_end_idx; map_task_idx+=STRIDE){
		
		keyval_arr_t *kv_arr_t = (keyval_arr_t *)&(kv_arr_t_arr[index]);
		index++;

		kv_arr_t->buff = buff;
		kv_arr_t->total_arr_len = total_arr_len;

		kv_arr_t->buff_len = 1024*100;
		kv_arr_t->buff_pos = 0;
		kv_arr_t->arr = NULL;
		kv_arr_t->arr_len = 0;
		
		//keyval_arr_t *kv_arr_p = &(d_g_state.d_intermediate_keyval_arr_arr[map_task_idx]);
		d_g_state.d_intermediate_keyval_arr_arr_p[map_task_idx] = kv_arr_t;
		
		/*(d_g_state.d_intermediate_keyval_arr_arr[map_task_idx]).buff = buff;
		(d_g_state.d_intermediate_keyval_arr_arr[map_task_idx]).buff_len = 1024*1024;
		(d_g_state.d_intermediate_keyval_arr_arr[map_task_idx]).buff_pos = 0;*/

		//void *val = d_g_state.d_input_keyval_arr[map_task_idx].val;
		char *key = (char *)(d_g_state.d_input_keys_shared_buff) + d_g_state.d_input_keyval_pos_arr[map_task_idx].keyPos;
		char *val = (char *)(d_g_state.d_input_vals_shared_buff) + d_g_state.d_input_keyval_pos_arr[map_task_idx].valPos;

		int valSize = d_g_state.d_input_keyval_pos_arr[map_task_idx].valSize;
		int keySize = d_g_state.d_input_keyval_pos_arr[map_task_idx].keySize;

		//printf("map_task_idx:%d keySize:%d  valSize:%d\n",map_task_idx, keySize, valSize);
		//printf("key:%d  keySize:%d\n",*(int *)key, keySize);
		//TODO calculate the key val pair here directly. 

		///////////////////////////////////////////////////////////
		
		gpu_map(key, val, keySize, valSize, &d_g_state, map_task_idx);
		///////////////////////////////////////////////////////////

	}//for

	//printf("task id:%d\n",TID);
	__syncthreads();
}//GPUMapPartitioner


int StartCPUMap2(thread_info_t* thread_info)
{		
	cpu_context *d_g_state = (cpu_context*)(thread_info->d_g_state);
	job_configuration *cpu_job_conf = (job_configuration*)(thread_info->job_conf);

	DoLog("there are %d input records for map tasks.",cpu_job_conf->num_input_record);
	if (cpu_job_conf->num_input_record<=0) { DoLog("Error: no any input keys"); exit(-1);}
	if (cpu_job_conf->input_keyval_arr == NULL) { DoLog("Error: input_keyval_arr == NULL"); exit(-1);}
	if (d_g_state->num_cpus_cores <= 0) {	DoLog("Error: d_g_state->num_cpus == 0"); exit(-1);}

	//-------------------------------------------------------
	//1, prepare buffer to store intermediate results
	//-------------------------------------------------------
	
	DoLog("prepare buffer to store intermediate results");
	
	keyval_arr_t *d_keyval_arr_p;
	int *count = NULL;
	
	//---------------------------------------------
	//3, determine the number of threads to run
	//---------------------------------------------

	DoLog("the number of cpus used in computation:%d",d_g_state->num_cpus_cores);
	
	//--------------------------------------------------
	//4, start map
	//--------------------------------------------------
	
	int num_threads = d_g_state->num_cpus_cores;
	DoLog("start CPUMapPartitioner num_threads:%d  num_input_record:%d",num_threads, cpu_job_conf->num_input_record);
	int num_records_per_thread = (cpu_job_conf->num_input_record+num_threads-1)/(num_threads);
	
	int start_idx = 0;
	int end_idx = 0;

	//pthread_t  *cpu_threads;
	//thread_info_t *cpu_threads_info;
	for (int tid = 0;tid<num_threads;tid++){
	
		end_idx = start_idx + num_records_per_thread;
		if (tid < (cpu_job_conf->num_input_record % num_threads) )
			end_idx++;
			
		d_g_state->panda_cpu_task_info[tid].start_idx = start_idx;
		if (end_idx > cpu_job_conf->num_input_record)
			end_idx = cpu_job_conf->num_input_record;
		d_g_state->panda_cpu_task_info[tid].end_idx = end_idx;
		
		if (pthread_create(&(d_g_state->panda_cpu_task[tid]),NULL,RunPandaCPUMapThread,(char *)&(d_g_state->panda_cpu_task_info[tid]))!=0) 
			perror("Thread creation failed!\n");
		start_idx = end_idx;
	}//for
	
	for (int tid = 0;tid<num_threads;tid++){
		void *exitstat;
		if (pthread_join(d_g_state->panda_cpu_task[tid],&exitstat)!=0) perror("joining failed");
	}//for
	
	DoLog("DONE *********** current intermediate len:%d", d_g_state->intermediate_keyval_arr_arr_p[0].arr_len);
	DoLog("DONE *********** current intermediate len:%d", d_g_state->intermediate_keyval_arr_arr_p[1].arr_len);
	DoLog("DONE *********** current intermediate len:%d", d_g_state->intermediate_keyval_arr_arr_p[2].arr_len);

	DoLog("CPU GROUP ID:[%d] DONE", thread_info->tid);
	return 0;
}//int 



int StartCPUMap(cpu_context *d_g_state)
{		
#ifdef ABC

	DoLog("there are %d map tasks.",d_g_state->num_input_record);
	if (d_g_state->num_input_record<=0) { DoLog("Error: no any input keys"); exit(-1);}
	if (d_g_state->input_keyval_arr == NULL) { DoLog("Error: input_keyval_arr == NULL"); exit(-1);}
	if (d_g_state->num_cpus_cores <= 0) {	DoLog("Error: d_g_state->num_cpus == 0"); exit(-1);}

	//-------------------------------------------------------
	//1, prepare buffer to store intermediate results
	//-------------------------------------------------------
	
	DoLog("prepare buffer to store intermediate results");
	
	keyval_arr_t *d_keyval_arr_p;
	int *count = NULL;
	
	//---------------------------------------------
	//3, determine the number of threads to run
	//---------------------------------------------

	DoLog("the number of cpus used in computation:%d",d_g_state->num_cpus_cores);
	
	//--------------------------------------------------
	//4, start map
	//--------------------------------------------------
	
	int num_threads = d_g_state->num_cpus_cores;

	DoLog("start CPUMapPartitioner num_threads:%d  num_input_record:%d",num_threads, d_g_state->num_input_record);
	int num_records_per_thread = (d_g_state->num_input_record+num_threads-1)/(num_threads);
	
	int start_idx = 0;
	int end_idx = 0;

	//pthread_t  *cpu_threads;
	//thread_info_t *cpu_threads_info;

	for (int tid = 0;tid<num_threads;tid++){
	
		end_idx = start_idx + num_records_per_thread;
		if (tid < (d_g_state->num_input_record % num_threads) )
			end_idx++;
			
		d_g_state->panda_cpu_task_info[tid].start_idx = start_idx;
		if (end_idx > d_g_state->num_input_record)
			end_idx = d_g_state->num_input_record;
		d_g_state->panda_cpu_task_info[tid].end_idx = end_idx;
		
		//pthread_t  *panda_cpu_task;
		//panda_cpu_task_info_t *panda_cpu_task_info;
		DoLog("tests");
		if (pthread_create(&(d_g_state->panda_cpu_task[tid]),NULL,RunPandaCPUMapThread,(char *)&(d_g_state->panda_cpu_task_info[tid]))!=0) 
			perror("Thread creation failed!\n");
		
		start_idx = end_idx;
	}//for

	
	for (int tid = 0;tid<num_threads;tid++){
		void *exitstat;
		if (pthread_join(d_g_state->panda_cpu_task[tid],&exitstat)!=0) perror("joining failed");
	}//for
	
	DoLog("DONE :%d tasks current intermediate len:%d",panda_cpu_task_info->end_idx - panda_cpu_task_info->start_idx, d_g_state->intermediate_keyval_arr_arr_p[0].arr_len);
	DoLog("DONE");
#endif
	return 0;

}//int 

//--------------------------------------------------
//StartGPUMap
//
//7/1/2012
//--------------------------------------------------
 

int StartGPUMap(gpu_context *d_g_state)
{		

	//-------------------------------------------------------
	//0, Check status of d_g_state;
	//-------------------------------------------------------

	DoLog("GPU_ID:[%d]  check num_input_record h_input_keyval_arr before StartGPUMap",d_g_state->gpu_id, d_g_state->num_input_record);
	if (d_g_state->num_input_record<0) { DoLog("Error: no any input keys"); exit(-1);}
	if (d_g_state->h_input_keyval_arr == NULL) { DoLog("Error: h_input_keyval_arr == NULL"); exit(-1);}
	if (d_g_state->num_mappers<=0) {d_g_state->num_mappers = (NUM_BLOCKS)*(NUM_THREADS);}
	if (d_g_state->num_reducers<=0) {d_g_state->num_reducers = (NUM_BLOCKS)*(NUM_THREADS);}

	//-------------------------------------------------------
	//1, prepare buffer to store intermediate results
	//-------------------------------------------------------
	//DoLog("prepare buffer to store intermediate results");

	keyval_arr_t *h_keyval_arr_arr = (keyval_arr_t *)malloc(sizeof(keyval_arr_t)*d_g_state->num_input_record);
	keyval_arr_t *d_keyval_arr_arr;
	checkCudaErrors(hipMalloc((void**)&(d_keyval_arr_arr),d_g_state->num_input_record*sizeof(keyval_arr_t)));
	
	for (int i=0; i<d_g_state->num_input_record;i++){
		h_keyval_arr_arr[i].arr = NULL;
		h_keyval_arr_arr[i].arr_len = 0;
	}//for
	//checkCudaErrors(hipMemcpy(d_keyval_arr_arr, h_keyval_arr_arr, sizeof(keyval_arr_t)*d_g_state->num_input_record,hipMemcpyHostToDevice));
	//d_g_state->d_intermediate_keyval_arr_arr = d_keyval_arr_arr;

	keyval_arr_t **d_keyval_arr_arr_p;
	checkCudaErrors(hipMalloc((void***)&(d_keyval_arr_arr_p),d_g_state->num_input_record*sizeof(keyval_arr_t*)));
	d_g_state->d_intermediate_keyval_arr_arr_p = d_keyval_arr_arr_p;
	
	int *count = NULL;
	checkCudaErrors(hipMalloc((void**)&(count),d_g_state->num_input_record*sizeof(int)));
	d_g_state->d_intermediate_keyval_total_count = count;
	checkCudaErrors(hipMemset(d_g_state->d_intermediate_keyval_total_count,0,d_g_state->num_input_record*sizeof(int)));

	//----------------------------------------------
	//3, determine the number of threads to run
	//----------------------------------------------
	//TODO determine the number of threads to run
	//DoLog("gpu_id[%d] determine the number of threads (NUM_BLOCKS, NUM_THREADS) to run GPUMapPartitioner",d_g_state->gpu_id);
	//int num_threads = d_g_state->num_input_record;
	//calculate NUM_BLOCKS, NUM_THREADS
	
	//--------------------------------------------------
	//4, start map
	//--------------------------------------------------
	/*dim3 h_dimBlock(512,1,1);
    dim3 h_dimGrid(4,1,1);
	dim3 h_dimThread(1,1,1);
	int sizeSmem = 128;*/
	//DoLog("start GPUMapPartitioner");
	hipDeviceSynchronize();
	
	//printf("avail_mem:%d \n",avail_mem);
	//modified for Panda Matrix Multiplication
	
	//int block_size = 10;
	//dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 grid(d_g_state->matrix_size / threads.x, d_g_state->matrix_size / threads.y);
	//dim3 grid(d_g_state->matrix_size / BLOCK_SIZE, d_g_state->matrix_size / BLOCK_SIZE);
	//DoLog(" matrix size :%d block_size:%d threads.y:%d threads.x:%d",d_g_state->matrix_size,  BLOCK_SIZE, threads.y, threads.x);
	int num_blocks = (d_g_state->num_mappers+(NUM_THREADS)-1)/(NUM_THREADS);
	//GPUMapPartitioner<<<NUM_BLOCKS,NUM_THREADS>>>(*d_g_state);
	DoLog("GPU_ID:[%d] NUM_BLOCKS:%d NUM_THREADS:%d to run GPUMapPartitioner",d_g_state->gpu_id, num_blocks, NUM_THREADS);
	GPUMapPartitioner<<<num_blocks,NUM_THREADS>>>(*d_g_state);
	
	hipDeviceSynchronize();

	//size_t total_mem,avail_mem, heap_limit;
	//checkCudaErrors(hipMemGetInfo( &avail_mem, &total_mem ));
	//printf("avail_mem:%d \n",avail_mem);
	DoLog("GPU_ID:[%d] DONE",d_g_state->gpu_id);
	return 0;
}//int 


//--------------------------------------------------
//start map
//
//1, get map input data on host
//2, upload map input data to device memory
//	 (keys, vals, keyOffsets, valOffsets, keySizes, valSizes)
//3, determine the number of threads to run 
//4, calculate intermediate data keys'buf size 
//	 and values' buf size
//5, do prefix sum on--
//	 i)		d_interKeysSizePerTask
//	 ii)	d_interValsSizePerTask
//	 iii)	d_interCountPerTask
//6, allocate intermediate memory on device memory
//7, start map
//8, free allocated memory
//--------------------------------------------------

/*
int startMap(Spec_t* spec, gpu_context *d_g_state)
{
	    
	Spec_t* g_spec = spec;

	if (g_spec->inputKeys == NULL) { DoLog("Error: no any input keys"); exit(0);}
	if (g_spec->inputVals == NULL) { DoLog("Error: no any input values"); exit(0); }
	if (g_spec->inputOffsetSizes == NULL) { DoLog( "Error: no any input pointer info"); exit(0); }
	if (g_spec->inputRecordCount == 0) {DoLog( "Error: invalid input record count"); exit(0);}
	
	//-------------------------------------------------------
	//1, get map input data on host
	//-------------------------------------------------------
	return 0;
}//return 0;
*/

void DestroyDGlobalState(gpu_context * d_g_state){
	
}//void 


void StartGPUShuffle(gpu_context * state){
	DoLog("GPU_ID:[%d] GPU Shuffle", state->gpu_id);
	gpu_context* d_g_state = state;

	Shuffle4GPUOutput(d_g_state);

	//DoLog("DONE");
}//void

void *RunPandaCPUMapThread(void *ptr){
		
	panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
	cpu_context *d_g_state = (cpu_context *)(panda_cpu_task_info->d_g_state); 
	job_configuration *cpu_job_conf = (job_configuration *)(panda_cpu_task_info->cpu_job_conf); 

	//DoLog("panda_cpu_task_info_t start_idx:%d end_idx:%d",panda_cpu_task_info->start_idx, panda_cpu_task_info->end_idx);
	for (int map_idx = panda_cpu_task_info->start_idx; map_idx < panda_cpu_task_info->end_idx; map_idx++){
		keyval_t *kv_p = (keyval_t *)(&(cpu_job_conf->input_keyval_arr[map_idx]));
		//void cpu_map(void *KEY, void*VAL, int keySize, int valSize, cpu_context *d_g_state, int map_task_idx){
		cpu_map(kv_p->key,kv_p->val,kv_p->keySize,kv_p->valSize,d_g_state,map_idx);
		//DoLog("finished map_task:%d at tid:%d",d_g_state->end_idx - d_g_state->start_idx, d_g_state->tid);
	}//for
	DoLog("DONE :%d tasks current intermediate len:%d",panda_cpu_task_info->end_idx - panda_cpu_task_info->start_idx, d_g_state->intermediate_keyval_arr_arr_p[0].arr_len);
	return NULL;
}

//Use Pthread to process Panda_Reduce
void * Panda_Reduce(void *ptr){
		
	thread_info_t *thread_info = (thread_info_t *)ptr;
		
	if(thread_info->device_type == GPU_ACC){
		
		int tid = thread_info->tid;
		int num_gpus = 1;//thread_info->num_gpus;
		
		if (num_gpus == 0){
			DoLog("thread_info->num_gpus == 0 return");
			return NULL;
		}//if
		
		hipSetDevice(tid % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
		int gpu_id;
		hipGetDevice(&gpu_id);
		gpu_context *d_g_state = (gpu_context *)(thread_info->d_g_state);
		DoLog( "start reduce tasks on GPU:%d tid:%d",gpu_id, tid);
		
		StartGPUReduce(d_g_state);
		
		
		}//if
		
	if(thread_info->device_type == CPU_ACC){
		
		cpu_context *d_g_state = (cpu_context *)(thread_info->d_g_state);
		DoLog("Start CPU Reduce Tasks");
		
		if (d_g_state->num_cpus_cores == 0){
			DoLog("d_g_state->num_cpus == 0 return");
			return NULL;
		}

		//StartCPUReduce(d_g_state);
		//panda_cpu_task_info_t *panda_cpu_task_info = (panda_cpu_task_info_t *)ptr;
		//cpu_context *d_g_state = (cpu_context *)(panda_cpu_task_info->d_g_state); 
		//DoLog("panda_cpu_task_info_t start_idx:%d end_idx:%d",panda_cpu_task_info->start_idx, panda_cpu_task_info->end_idx);

		for (int map_idx = 0; map_idx < d_g_state->sorted_keyvals_arr_len; map_idx++){

		keyvals_t *kv_p = (keyvals_t *)(&(d_g_state->sorted_intermediate_keyvals_arr[map_idx]));
		cpu_reduce(kv_p->key, kv_p->vals, kv_p->keySize, kv_p->val_arr_len, d_g_state);

		}//for
		DoLog("DONE");
		
	}//if	
	
	//hipFree(d_filebuf);
	//handle the buffer different
	//free(h_filebuf);
	//handle the buffer different
	return NULL;
}//void

//--------------------------------------------------------
//get a value from value list of the same key
//
//param	: vals
//param	: interOffsetSizes
//param	: index
//return: the wanted value
//--------------------------------------------------------

__device__ void *GetVal(void *vals, int4* interOffsetSizes, int keyIndex, int valStartIndex)
{
	int4 offset = interOffsetSizes[valStartIndex];
	return (void*)((char*)vals + keyIndex * offset.w);
}

__device__ void *GetKey(void *key, int4* interOffsetSizes, int keyIndex, int valStartIndex)
{
	int4 offset = interOffsetSizes[valStartIndex];
	return (void*)((char*)key + keyIndex * offset.y);
}

//-------------------------------------------------------
//Reducer
//-------------------------------------------------------



__global__ void ReducePartitioner(gpu_context d_g_state)
{
	int num_records_per_thread = (d_g_state.d_sorted_keyvals_arr_len+(gridDim.x*blockDim.x)-1)/(gridDim.x*blockDim.x);
	int block_start_idx = num_records_per_thread*blockIdx.x*blockDim.x;

	int thread_start_idx = block_start_idx 
		+ (threadIdx.x/STRIDE)*num_records_per_thread*STRIDE
		+ (threadIdx.x%STRIDE);

	int thread_end_idx = thread_start_idx+num_records_per_thread*STRIDE;
	if(thread_end_idx>d_g_state.d_sorted_keyvals_arr_len)
		thread_end_idx = d_g_state.d_sorted_keyvals_arr_len;

	//printf("ReducePartitioner: TID:%d  start_idx:%d  end_idx:%d d_sorted_keyvals_arr_len:%d\n",TID,thread_start_idx,thread_end_idx,d_g_state.d_sorted_keyvals_arr_len);

	int start, end;
	for(int reduce_task_idx=thread_start_idx; reduce_task_idx < thread_end_idx; reduce_task_idx+=STRIDE){
		if (reduce_task_idx==0)
			start = 0;
		else
			start = d_g_state.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx-1];
		end = d_g_state.d_pos_arr_4_sorted_keyval_pos_arr[reduce_task_idx];

		val_t *val_t_arr = (val_t*)malloc(sizeof(val_t)*(end-start));
		//assert(val_t_arr!=NULL);
		int keySize = d_g_state.d_keyval_pos_arr[start].keySize;
		int keyPos = d_g_state.d_keyval_pos_arr[start].keyPos;
		void *key = (char*)d_g_state.d_sorted_keys_shared_buff+keyPos;
		//printf("keySize;%d keyPos:%d key:%s\n",keySize,keyPos,key);
		//printf("reduce_task_idx:%d		keyPos:%d,  keySize:%d, key:% start:%d end:%d\n",reduce_task_idx,keyPos,keySize,start,end);
		//printf("start:%d end:%d\n",start,end);
		
		
		for (int index = start;index<end;index++){
			int valSize = d_g_state.d_keyval_pos_arr[index].valSize;
			int valPos = d_g_state.d_keyval_pos_arr[index].valPos;
			val_t_arr[index-start].valSize = valSize;
			val_t_arr[index-start].val = (char*)d_g_state.d_sorted_vals_shared_buff + valPos;
		}   //for
		gpu_reduce(key, val_t_arr, keySize, end-start, d_g_state);

	}//for
}



//----------------------------------------------
//start reduce
//
//1, if there is not a reduce phase, just return
//   then user uses spec->interKeys/spec->intervals 
//   for further processing
//2, get reduce input data on host
//3, upload reduce input data onto device memory
//4, determine the number of threads to run
//5, calculate output data keys'buf size 
//	 and values' buf size
//6, do prefix sum on--
//	 i)		d_outputKeysSizePerTask
//	 ii)	d_outputValsSizePerTask
//	 iii)	d_outputCountPerTask
//7, allocate output memory on device memory
//8, start reduce
//9, copy output data to Spect_t structure
//10,free allocated memory
//----------------------------------------------
		
void StartGPUReduce(gpu_context *d_g_state)
{	
	hipDeviceSynchronize(); 
	d_g_state->d_reduced_keyval_arr_len = d_g_state->d_sorted_keyvals_arr_len;
	checkCudaErrors(hipMalloc((void **)&(d_g_state->d_reduced_keyval_arr), sizeof(keyval_t)*d_g_state->d_reduced_keyval_arr_len));
	
	DoLog("number of reduce tasks:%d",d_g_state->d_sorted_keyvals_arr_len);
	hipDeviceSynchronize(); 
	

	int num_blocks = (d_g_state->num_reducers+(NUM_THREADS)-1)/(NUM_THREADS);
	DoLog("num_blocks:%d NUM_THREADS:%d",num_blocks,NUM_THREADS);
	ReducePartitioner<<<num_blocks,NUM_THREADS>>>(*d_g_state);
	hipDeviceSynchronize(); 

	DoLog("DONE");
}//void

/*void *Panda_CPU_Map(void *ptr){
return NULL;
}*/

void* Panda_Map(void *ptr){
		
	//DoLog("panda_map");
	thread_info_t *thread_info = (thread_info_t *)ptr;
		
	if(thread_info->device_type == GPU_ACC){
		
		gpu_context *d_g_state = (gpu_context *)(thread_info->d_g_state);
		//DoLog("gpu_id in current Panda_Map:%d",gpu_id);
		InitGPUDevice(thread_info);
		
		DoLog("GPU_ID:[%d] Init GPU MapReduce Load Data From Host to GPU memory",d_g_state->gpu_id);
		InitGPUMapReduce3(d_g_state);
		//printData2<<<1,1>>>(*d_g_state);

		DoLog("GPU_ID:[%d] Start GPU Map Tasks",d_g_state->gpu_id);
		StartGPUMap(d_g_state);
		
		//d_g_state->d_intermediate_keyval_total_count;
		//checkCudaErrors(hipMemset(d_g_state->d_intermediate_keyval_total_count,0,d_g_state->num_input_record*sizeof(int)));
		
		StartGPUShuffle(d_g_state);
		
	}//if
		
	if(thread_info->device_type == CPU_ACC){

		//DoLog("CPU_ACC");
		cpu_context *d_g_state = (cpu_context *)(thread_info->d_g_state);
		DoLog("CPU_GROUP_ID:[%d] Init CPU Device",d_g_state->cpu_group_id);
		InitCPUDevice(thread_info);
		
		//DoLog("Init CPU MapReduce");
		InitCPUMapReduce2(thread_info);

		DoLog("CPU_GROUP_ID:[%d] Start CPU Map Tasks",d_g_state->cpu_group_id);
		StartCPUMap2(thread_info);
		//d_g_state->d_intermediate_keyval_total_count;
		//checkCudaErrors(hipMemset(d_g_state->d_intermediate_keyval_total_count,0,d_g_state->num_input_record*sizeof(int)));
		StartCPUShuffle2(thread_info);
		
	}	
			
	return NULL;
}//FinishMapReduce2(d_g_state);


void FinishMapReduce(Spec_t* spec)
{
	DoLog( "=====finish map/reduce=====");
}//void


void FinishMapReduce2(gpu_context* state)
{
	
	size_t total_mem,avail_mem, heap_limit;
	checkCudaErrors(hipMemGetInfo( &avail_mem, &total_mem ));

	DoLog("avail_mem:%d",avail_mem);

	
}//void





#endif //__PANDALIB_CU__